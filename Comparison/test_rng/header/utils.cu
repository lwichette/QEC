#include "hip/hip_runtime.h"
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <time.h>
#include <hiprand.h>
#include <hipcub/hipcub.hpp>
#include <thrust/complex.h>
#include <cmath>
#include <math.h>
#include <vector>
#include <string>
#include <sys/stat.h>
#include <unistd.h>
#include <filesystem>
#include "defines.h"
#include "utils.cuh"
#include "cudamacro.h"

using namespace std;

void *d_temp_nx = NULL;
size_t temp_storage_nx = 0;

void *d_temp_nx_thrust = NULL;
size_t temp_storage_nx_thrust = 0;

void *d_temp_nis = NULL;
size_t temp_storage_nis = 0;

void *d_temp_nie = NULL;
size_t temp_storage_nie = 0;

// Initialize lattice spins
__global__ void init_spins_up(
    signed char* lattice, const long long nx, const long long ny, const int num_lattices
){
    const long long  tid = static_cast<long long>(blockDim.x) * blockIdx.x + threadIdx.x;
    if (tid >= nx * ny * num_lattices) return;

    lattice[tid] = 1;
}

__global__ void init_randombond(
    signed char* interactions, const float* __restrict__ interaction_randvals,
    const long long nx, const long long ny, const int num_lattices, const float p
){

        const long long tid = static_cast<long long>(threadIdx.x + blockIdx.x * blockDim.x);

        if (tid >= 2*nx*ny*num_lattices) return;

        float bondrandval = interaction_randvals[tid];
        signed char bondval = (bondrandval<p)? -1 : 1;
        interactions[tid] = bondval;
}

// Initialize lattice spins
__global__ void init_spins(
    signed char* lattice, const float* __restrict__ randvals,
    const long long nx, const long long ny, const int num_lattices
){
        const long long  tid = static_cast<long long>(blockDim.x * blockIdx.x + threadIdx.x);
        if (tid >= nx * ny * num_lattices) return;

        float randval = randvals[tid];
        signed char val = (randval < 0.5f) ? -1 : 1;
        lattice[tid] = val;
}

void init_interactions_with_seed(
    signed char* interactions, hiprandGenerator_t interaction_rng, float* interaction_randvals,
    const long long nx, const long long ny, const int num_lattices, const float p, const int blocks
){
    CHECK_CURAND(hiprandGenerateUniform(interaction_rng,interaction_randvals, num_lattices*nx*ny*2));
    init_randombond<<<blocks, THREADS>>>(interactions, interaction_randvals, nx, ny, num_lattices, p);
}

void initialize_spins(
    signed char* lattice_b, signed char* lattice_w, hiprandGenerator_t lattice_rng, float* lattice_randvals,
    const long long nx, const long long ny, const int num_lattices, bool up, const int blocks, bool read_lattice, std::string filename_b, std::string filename_w
){

    if (read_lattice){
        // Initialization of black lattice
        if (std::filesystem::exists(filename_b.c_str())){

            std::vector<signed char> charVector_b;

            // black lattice read
            std::ifstream inFile_b(filename_b);

            int value;

            if (!inFile_b.is_open()) {
                std::cerr << "Error opening file for reading." << std::endl;
                return;  // Return an empty vector in case of an error
            }

            // Read each value from the file
            while (inFile_b >> value) {
                // Reverse the mapping: 0 to -1 and 1 to 1
                charVector_b.push_back((value == 0) ? -1 : 1);
            }
            // Close the file
            inFile_b.close();

            cout << "Initialized black lattice with preceeding results. ";
            // // printing initialized elements
            // for (const auto& element : charVector_b) {
            //     cout << static_cast<int>(element) << " ";
            // }
            cout << endl;

            CHECK_CUDA(hipMemcpy(lattice_b, charVector_b.data(), num_lattices * nx * ny /2 * sizeof(*lattice_b), hipMemcpyHostToDevice));
        }
        else{
            std::cerr << "Error opening file for reading. Could not find black lattice file." << std::endl;
            return;  // Return an empty vector in case of an error
        }

        // Initialization of white lattice
        if (std::filesystem::exists(filename_w)){

            std::vector<signed char> charVector_w;

            // black lattice read
            std::ifstream inFile_w(filename_w);

            int value;

            if (!inFile_w.is_open()) {
                std::cerr << "Error opening file for reading." << std::endl;
                return;  // Return an empty vector in case of an error
            }

            // Read each value from the file
            while (inFile_w >> value) {
                // Reverse the mapping: 0 to -1 and 1 to 1
                charVector_w.push_back((value == 0) ? -1 : 1);
            }

            // Close the file
            inFile_w.close();

            cout << "Initialized white lattice with preceeding results. ";
            // // printing initialized elements
            // for (const auto& element : charVector_w) {
            //     cout << static_cast<int>(element) << " ";
            // }
            cout << endl;

            CHECK_CUDA(hipMemcpy(lattice_w, charVector_w.data(), num_lattices * nx * ny /2 * sizeof(*lattice_w), hipMemcpyHostToDevice));
        }
        else{
            std::cerr << "Error opening file for reading. Could not find white lattice file." << std::endl;
            return;  // Return an empty vector in case of an error
        }
    }
    else {
        if (up){
            init_spins_up<<<blocks,THREADS>>>(lattice_b, nx, ny/2, num_lattices);
            init_spins_up<<<blocks,THREADS>>>(lattice_w, nx, ny/2, num_lattices);
        }
        else{
            //Initialize the arrays for white and black lattice
            CHECK_CURAND(hiprandGenerateUniform(lattice_rng, lattice_randvals, nx*ny/2*num_lattices));
            init_spins<<<blocks, THREADS>>>(lattice_b, lattice_randvals, nx, ny/2, num_lattices);
            //Initialize the arrays for white and black lattice
            CHECK_CURAND(hiprandGenerateUniform(lattice_rng, lattice_randvals, nx*ny/2*num_lattices));
            init_spins<<<blocks, THREADS>>>(lattice_w, lattice_randvals, nx, ny/2, num_lattices);
        }
    }
}

void write_updated_lattices(signed char *lattice_b, signed char *lattice_w, const long long nx, const long long ny, const int num_lattices, std::string lattice_b_file_name, std::string lattice_w_file_name){
    printf("Writing updated lattices to %s and %s \n", lattice_b_file_name.c_str(), lattice_w_file_name.c_str());
    // copy to host
    std::vector<signed char> h_lattice_b(num_lattices * nx * ny / 2);
    std::vector<signed char> h_lattice_w(num_lattices * nx * ny / 2);
    CHECK_CUDA(hipMemcpy(h_lattice_b.data(), lattice_b, num_lattices * nx * ny / 2 * sizeof(*lattice_b), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_lattice_w.data(), lattice_w, num_lattices * nx * ny / 2 * sizeof(*lattice_w), hipMemcpyDeviceToHost));
    // Open a file for writing the black lattice
    std::ofstream outFile_b(lattice_b_file_name, std::ios::out | std::ios::binary | std::ios::trunc);
    // Check if the file is open
    if (!outFile_b.is_open()) {
        std::cerr << "Error opening file for writing." << std::endl;
        return;
    }
    // Write each element of the black lattice vector to the file
    for (const auto& element : h_lattice_b) {
        char mappedValue = (element == 1) ? '1' : '0';
        outFile_b << mappedValue;
    }
    // Close the black lattice file
    outFile_b.close();
    // Open a file for writing the white lattice
    std::ofstream outFile_w(lattice_w_file_name, std::ios::out | std::ios::binary | std::ios::trunc);
    // Check if the file is open
    if (!outFile_w.is_open()) {
        std::cerr << "Error opening file for writing." << std::endl;
        return;
    }
    // Write each element of the white lattice vector to the file
    for (const auto& element : h_lattice_w) {
        char mappedValue = (element == 1) ? '1' : '0';
        outFile_w << mappedValue;
    }
    // Close the white lattice file
    outFile_w.close();
}

void write_lattice_to_disc(signed char *lattice_b, signed char *lattice_w, std::string filename, const long long nx, const long long ny, const int num_lattices) {
    printf("Writing lattice to %s...\n", filename.c_str());

    std::vector<signed char> lattice_h(nx*ny);
    std::vector<signed char> lattice_w_h(nx*ny/2);
    std::vector<signed char> lattice_b_h(nx*ny/2);

    hipMemcpy(lattice_b_h.data(), lattice_b, nx * ny/2 * sizeof(*lattice_b), hipMemcpyDeviceToHost);
    hipMemcpy(lattice_w_h.data(), lattice_w, nx * ny/2 * sizeof(*lattice_w), hipMemcpyDeviceToHost);

    int offset;

    for (int l = 0; l < num_lattices; l++){

        offset = l*nx*ny/2;

        for (int i = 0; i < nx; i++){
            for (int j=0; j < ny/2; j++){
                if (i%2 == 0){
                    lattice_h[i*ny+2*j+1] = lattice_w_h[offset + i*ny/2+j];
                    lattice_h[i*ny+2*j] = lattice_b_h[offset + i*ny/2+j];
                }
                else{
                    lattice_h[i*ny+2*j] = lattice_w_h[offset + i*ny/2+j];
                    lattice_h[i*ny+2*j+1] = lattice_b_h[offset + i*ny/2+j];
                }
            }
        }

        std::ofstream f;
        f.open(filename + std::string(".txt"));

        if (f.is_open()) {
            for (int i = 0; i < nx; i++) {
                for (int j = 0; j < ny; j++) {
                    f << (int)lattice_h[i * ny + j] << " ";
                }
                f << std::endl;
            }
        }
        f.close();
    }
}

void write_bonds(signed char* interactions, std::string filename, const long nx, const long ny, const int num_lattices){
    printf("Writing bonds to %s ...\n", filename.c_str());

    std::vector<signed char> interactions_host(2*nx*ny*num_lattices);

    CHECK_CUDA(hipMemcpy(interactions_host.data(),interactions, 2*num_lattices*nx*ny*sizeof(*interactions), hipMemcpyDeviceToHost));

    int offset;

    for (int l=0; l<num_lattices; l++){

        offset = l*nx*ny*2;

        std::ofstream f;
        f.open(filename + std::to_string(l) + std::string(".txt"));
        if (f.is_open()) {
            for (int i = 0; i < 2*nx; i++) {
                for (int j = 0; j < ny; j++) {
                    f << (int)interactions_host[offset + i * ny + j] << " ";
                }
                f << std::endl;
            }
        }
        f.close();
    }
}

template<bool is_black>
__global__ void update_lattice(
    signed char* lattice, signed char* __restrict__ op_lattice, const float* __restrict__ randvals, const signed char* interactions,
    const double *inv_temp, const long long nx, const long long ny, const int num_lattices, double* d_energy
) {

    const long long tid = static_cast<long long>(blockDim.x)*blockIdx.x + threadIdx.x;

    if (tid >= nx*ny*num_lattices) return;

    // Calculate in which lattice we are
    int l_id = tid/(nx*ny);

    // Project tid back to single lattice
    int tid_sl = tid - l_id*nx*ny;

    int i = tid_sl/ny;
    int j = tid_sl%ny;

    // Set up periodic boundary conditions
    int ipp = (i + 1 < nx) ? i + 1 : 0;
    int inn = (i - 1 >= 0) ? i - 1: nx - 1;
    int jpp = (j + 1 < ny) ? j + 1 : 0;
    int jnn = (j - 1 >= 0) ? j - 1: ny - 1;

    int joff;
    int jcouplingoff;
    int icouplingpp;
    int icouplingnn;

    int offset = l_id * nx * ny;
    int offset_i = l_id * nx * ny * 4;

    if (!is_black) {
        icouplingpp = offset_i + 2*(nx-1)*ny + 2*(ny*(i+1) + j) + (i+1)%2;
        icouplingnn = offset_i + 2*(nx-1)*ny + 2*(ny*(inn+1) + j) + (i+1)%2;

        joff = (i % 2) ? jnn : jpp;

        if (i % 2) {
            jcouplingoff = offset_i + 2 * (i * ny + joff) + 1;
        } else {
            if (j + 1 >= ny) {
                jcouplingoff = offset_i + 2 * (i * ny + j + 1) - 1;
            } else {
                jcouplingoff = offset_i + 2 * (i * ny + joff) - 1;
            }
        }
    } else {
        icouplingpp = offset_i + 2*(nx-1)*ny + 2*(ny*(i+1) + j) + i%2;
        icouplingnn = offset_i + 2*(nx-1)*ny + 2*(ny*(inn+1) + j) + i%2;

        joff = (i % 2) ? jpp : jnn;

        if (i % 2) {
            if (j+1 >= ny) {
                jcouplingoff = offset_i + 2 * (i * ny + j + 1) - 1;
            } else {
                jcouplingoff = offset_i + 2 * (i * ny + joff) - 1;
            }
        } else {
            jcouplingoff = offset_i + 2 * (i * ny + joff) + 1;
        }
    }

    // Compute sum of nearest neighbor spins times the coupling
    signed char nn_sum = op_lattice[offset + inn*ny + j]*interactions[icouplingnn] + op_lattice[offset + i*ny + j]*interactions[offset_i + 2*(i*ny + j)]
                        + op_lattice[offset + ipp*ny + j]*interactions[icouplingpp] + op_lattice[offset + i*ny + joff]*interactions[jcouplingoff];


    signed char lij = lattice[offset + i*ny + j];

    // set device energy for each temp and each spin on lattice
    d_energy[tid]=inv_temp[l_id]*nn_sum*lij;

    // Determine whether to flip spin
    float acceptance_ratio = exp(-2 * d_energy[tid]);
    if (randvals[offset + i*ny + j] < acceptance_ratio) {
        lattice[offset + i*ny + j] = -lij;
        d_energy[tid] *= -1;
    }
}

void update(
    signed char *lattice_b, signed char *lattice_w, float* randvals, hiprandGenerator_t rng, const signed char* interactions,
    const double *inv_temp, const long long nx, const long long ny, const int num_lattices, const int blocks, double *d_energy
) {

    // Update black
    CHECK_CURAND(hiprandGenerateUniform(rng, randvals, num_lattices*nx*ny/2));
    update_lattice<true><<<blocks, THREADS>>>(lattice_b, lattice_w, randvals, interactions, inv_temp, nx, ny/2, num_lattices, d_energy);

    // Update white
    CHECK_CURAND(hiprandGenerateUniform(rng, randvals, num_lattices*nx*ny/2));
    update_lattice<false><<<blocks, THREADS>>>(lattice_w, lattice_b, randvals, interactions, inv_temp, nx, ny/2, num_lattices, d_energy);
}

__global__ void B2_lattices(
    signed char *lattice_b, signed char *lattice_w, const double *wave_vector,
    thrust::complex<double> *sum, const int nx, const int ny, const int num_lattices
){

    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= nx*ny*num_lattices) return;

    int lattice_id = tid/(nx*ny);
    int offset = lattice_id*nx*ny;
    int tid_single_lattice = tid - offset;

    int i = tid_single_lattice/ny;
    int j = tid_single_lattice%ny;

    int b_orig_j;
    int w_orig_j;

    if (i%2==0){
        b_orig_j = 2*j +1;
        w_orig_j = 2*j;
    }

    else{
        b_orig_j = 2*j;
        w_orig_j = 2*j + 1;
    }

    thrust::complex<float> imag = thrust::complex<float>(0, 1.0f);

    float dot_b = wave_vector[0]*i + wave_vector[1]*b_orig_j;
    float dot_w = wave_vector[0]*i + wave_vector[1]*w_orig_j;

    sum[tid] = lattice_b[tid]*exp(imag*dot_b) + lattice_w[tid]*exp(imag*dot_w);
}

void calculate_B2(
    thrust::complex<double> *d_sum, signed char *lattice_b, signed char *lattice_w, thrust::complex<double> *d_store_sum, const double *d_wave_vector, const long nx, const long ny, const int num_lattices, const int blocks
){
    // Calculate B2 and reduce sum
    B2_lattices<<<blocks, THREADS>>>(lattice_b, lattice_w, d_wave_vector, d_sum, nx, ny/2, num_lattices);

    // cant we parallalize this too?
    for (int i=0; i<num_lattices; i++){

        if (temp_storage_nx_thrust == 0){
            CHECK_CUDA(hipcub::DeviceReduce::Sum(d_temp_nx_thrust, temp_storage_nx_thrust, d_sum + i*nx*ny/2, &d_store_sum[i], nx*ny/2));
            CHECK_CUDA(hipMalloc(&d_temp_nx_thrust, temp_storage_nx_thrust));
        }

        CHECK_CUDA(hipcub::DeviceReduce::Sum(d_temp_nx_thrust, temp_storage_nx_thrust, d_sum + i*nx*ny/2, &d_store_sum[i], nx*ny/2));
    }
}

__global__ void abs_square(thrust::complex<double> *d_store_sum, const int num_lattices){

    const long long tid = static_cast<long long>(blockDim.x)*blockIdx.x + threadIdx.x;

    if (tid >= num_lattices) return;

    d_store_sum[tid] = thrust::abs(d_store_sum[tid]) * thrust::abs(d_store_sum[tid]);
}

int create_results_folder(char* results){
    struct stat sb;

    if (stat(results, &sb) == 0){
        std::cout << "Results already exist, check file name";
        return 0;
    }
    else{
        mkdir(results, S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
        return 1;
    }
}

template<bool is_black>
__global__ void update_lattice_ob(
    signed char* lattice, signed char* __restrict__ op_lattice, const float* __restrict__ randvals, const signed char* interactions,
    const double *inv_temp, const long long nx, const long long ny, const int num_lattices, double* d_energy
){

    const long long tid = static_cast<long long>(blockDim.x)*blockIdx.x + threadIdx.x;

    if (tid >= nx*ny*num_lattices) return;

    // Calculate in which lattice we are
    int l_id = tid/(nx*ny);

    // Project tid back to single lattice
    int tid_sl = tid - l_id*nx*ny;

    int i = tid_sl/ny;
    int j = tid_sl%ny;

    // Set up periodic boundary conditions
    int ipp = (i + 1 < nx) ? i + 1 : 0;
    int inn = (i - 1 >= 0) ? i - 1: nx - 1;
    int jpp = (j + 1 < ny) ? j + 1 : 0;
    int jnn = (j - 1 >= 0) ? j - 1: ny - 1;

    int joff;
    int jcouplingoff;
    int icouplingpp;
    int icouplingnn;

    int offset = l_id * nx * ny;
    int offset_i = l_id * nx * ny * 4;

    int c_up = 1-inn/(nx-1);
    int c_down = 1-(i+1)/nx;
    int c_side;

    if (!is_black) {
        icouplingpp = offset_i + 2*(nx-1)*ny + 2*(ny*(i+1) + j) + (i+1)%2;
        icouplingnn = offset_i + 2*(nx-1)*ny + 2*(ny*(inn+1) + j) + (i+1)%2;

        joff = (i % 2) ? jnn : jpp;

        if (i % 2) {
            
            jcouplingoff = offset_i + 2 * (i * ny + joff) + 1;

            c_side = 1 - jnn/(ny-1);

        } else {
            
            c_side = 1 - (j+1)/ny;

            if (j + 1 >= ny) {
                jcouplingoff = offset_i + 2 * (i * ny + j + 1) - 1;
            } 
            else {
                jcouplingoff = offset_i + 2 * (i * ny + joff) - 1;
            }
        }
    }
    else {

        icouplingpp = offset_i + 2*(nx-1)*ny + 2*(ny*(i+1) + j) + i%2;
        icouplingnn = offset_i + 2*(nx-1)*ny + 2*(ny*(inn+1) + j) + i%2;

        joff = (i % 2) ? jpp : jnn;

        if (i % 2) {
            
            c_side = 1-(j+1)/ny;

            if (j+1 >= ny) {
                jcouplingoff = offset_i + 2 * (i * ny + j + 1) - 1;
            } 
            else {
                jcouplingoff = offset_i + 2 * (i * ny + joff) - 1;
            }
        } 
        else {
            c_side = 1-jnn/(ny-1);
            jcouplingoff = offset_i + 2 * (i * ny + joff) + 1;
        }
    }

    signed char nn_sum = op_lattice[offset + inn*ny + j]*interactions[icouplingnn]*c_up + op_lattice[offset + i*ny + j]*interactions[offset_i + 2*(i*ny + j)]
                        + op_lattice[offset + ipp*ny + j]*interactions[icouplingpp]*c_down + op_lattice[offset + i*ny + joff]*interactions[jcouplingoff]*c_side;

    // Determine whether to flip spin

    // The exponent is exactly what calc_energy_ob does and which is calles again to store energy over same iterator in update loop. Instead here should be filled the energy array directly

    signed char lij = lattice[offset + i*ny + j];

    // set device energy for each temp and each spin on lattice
    d_energy[tid]=inv_temp[l_id]*nn_sum*lij;

    float acceptance_ratio = exp(-2*d_energy[tid]);
    if (randvals[offset + i*ny + j] < acceptance_ratio) {
        lattice[offset + i*ny + j] = -lij;
        d_energy[tid] *= -1;
    }
}

void update_ob(
    signed char *lattice_b, signed char *lattice_w, float* randvals, hiprandGenerator_t rng, const signed char* interactions,
    const double *inv_temp, const long long nx, const long long ny, const int num_lattices, const int blocks, double *d_energy
) {

    // Update black and store final Hamiltonian value for each cross term in d_energy
    CHECK_CURAND(hiprandGenerateUniform(rng, randvals, num_lattices*nx*ny/2));
    update_lattice_ob<true><<<blocks, THREADS>>>(lattice_b, lattice_w, randvals, interactions, inv_temp, nx, ny/2, num_lattices, d_energy);

    // Update white and store final Hamiltonian value for each cross term in d_energy
    CHECK_CURAND(hiprandGenerateUniform(rng, randvals, num_lattices*nx*ny/2));
    update_lattice_ob<false><<<blocks, THREADS>>>(lattice_w, lattice_b, randvals, interactions, inv_temp, nx, ny/2, num_lattices, d_energy);
}

__global__ void incrementalSumMagnetization(thrust::complex<double> *d_store_sum_0, thrust::complex<double> *d_store_sum_k, const int num_lattices, double *d_storeIncrementalSumMag_0, double *d_storeIncrementalSumMag_k){

    const long long tid = static_cast<long long>(blockDim.x)*blockIdx.x + threadIdx.x;

    if (tid >= num_lattices) return;

    d_storeIncrementalSumMag_0[tid] += d_store_sum_0[tid].real();
    d_storeIncrementalSumMag_k[tid] += d_store_sum_k[tid].real();

}