#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <time.h>
#include <hiprand.h>
#include <hipcub/hipcub.hpp>
#include <thrust/complex.h>
#include <math.h>
#include <vector>
#include <string>
#include <sys/stat.h>

#include "../header/utils.cuh"
#include "../header/defines.h"

using namespace std;

int main(void){

    // Initialize all possible parameters
    int num_iterations_seeds = 100;
    int num_iterations_error = 100;

    int niters = 1000;
    int nwarmup = 100;

    int nx = 14;
    int ny = 14;

    float prob = 0.085;
    float coupling_constant = 1.0f/2.0f*log((1-prob)/prob);
    float inv_temp = coupling_constant;

    unsigned long long seeds_spins = 0ULL;
    unsigned long long seeds_interactions = 0ULL;

    // Variables used for sum reduction
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    // Allocate the wave vectors and copy it to GPU memory
    std::array<float, 2> wave_vector_0 = {0,0};

    float *d_wave_vector_0;
    hipMalloc(&d_wave_vector_0, 2 * sizeof(*d_wave_vector_0));
    hipMemcpy(d_wave_vector_0, wave_vector_0.data(), 2*sizeof(float), hipMemcpyHostToDevice);

    int blocks = (nx*ny*2 + THREADS -1)/THREADS;

    std::array<float, 2> wave_vector_k = {2.0*M_PI/nx,0};

    float *d_wave_vector_k;
    hipMalloc(&d_wave_vector_k, 2 * sizeof(*d_wave_vector_k));
    hipMemcpy(d_wave_vector_k, wave_vector_k.data(), 2*sizeof(float), hipMemcpyHostToDevice);

    auto t0 = std::chrono::high_resolution_clock::now();

    // Weighted error
    float *d_error_weight_0, *d_error_weight_k;
    hipMalloc(&d_error_weight_0, num_iterations_error*sizeof(*d_error_weight_0));
    hipMalloc(&d_error_weight_k, num_iterations_error*sizeof(*d_error_weight_k));

    // Loop over different errors
    for (int j=0; j < num_iterations_error; j++){

        // Initialize arrays on the GPU to store results per spin system for energy and sum of B2
        thrust::complex<float> *d_store_sum_0, *d_store_sum_k;
        hipMalloc(&d_store_sum_0, num_iterations_seeds*sizeof(*d_store_sum_0));
        hipMalloc(&d_store_sum_k, num_iterations_seeds*sizeof(*d_store_sum_k));

        float *d_store_energy;
        hipMalloc(&d_store_energy, num_iterations_seeds*sizeof(*d_store_energy));

        //Setup interaction lattice on device
        signed char *d_interactions;
        hipMalloc(&d_interactions, nx*ny*2*sizeof(*d_interactions));

        init_interactions_with_seed(d_interactions, seeds_interactions, nx, ny, prob);

        // Loop over number of iterations
        for (int i=0; i<num_iterations_seeds; i++){

            // Setup black and white lattice arrays on device
            signed char *lattice_b, *lattice_w;
            hipMalloc(&lattice_b, nx * ny/2 * sizeof(*lattice_b));
            hipMalloc(&lattice_w, nx * ny/2 * sizeof(*lattice_w));

            init_spins_with_seed(lattice_b, lattice_w, seeds_spins, nx, ny);

            // Setup cuRAND generator
            hiprandGenerator_t rng;
            hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
            hiprandSetPseudoRandomGeneratorSeed(rng, seeds_spins);

            float *randvals;
            hipMalloc(&randvals, nx * ny/2 * sizeof(*randvals));

            hipDeviceSynchronize();

            // Warmup iterations
            //printf("Starting warmup...\n");
            for (int j = 0; j < nwarmup; j++) {
                update(lattice_b, lattice_w, randvals, rng, d_interactions, inv_temp, nx, ny, coupling_constant);
            }

            hipDeviceSynchronize();

            for (int j = 0; j < niters; j++) {
                update(lattice_b, lattice_w, randvals, rng, d_interactions, inv_temp, nx, ny,coupling_constant);
                //if (j % 1000 == 0) printf("Completed %d/%d iterations...\n", j+1, niters);
            }

            hipDeviceSynchronize();

            calculate_B2(lattice_b, lattice_w, d_store_sum_0, d_wave_vector_0, i, nx, ny);
            calculate_B2(lattice_b, lattice_w, d_store_sum_k, d_wave_vector_k, i, nx, ny);

            calculate_energy(lattice_b, lattice_w, d_interactions, d_store_energy, coupling_constant, i, nx, ny);

            seeds_spins += 1;

            //write_lattice(lattice_b, lattice_w, "lattice/final_lattice_" + std::to_string(i) + ".txt", nx, ny);

            hipFree(lattice_b);
            hipFree(lattice_w);
            hipFree(randvals);
            hiprandDestroyGenerator(rng);
        }

        // Take absolute square + exp
        abs_square<<<blocks, THREADS>>>(d_store_sum_0, num_iterations_seeds);
        abs_square<<<blocks, THREADS>>>(d_store_sum_k, num_iterations_seeds);

        exp_beta<<<blocks, THREADS>>>(d_store_energy, inv_temp, num_iterations_seeds, nx);

        // Calculate partition function
        float *d_partition_function;
        hipMalloc(&d_partition_function, sizeof(float));

        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_store_energy, d_partition_function, num_iterations_seeds);
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_store_energy, d_partition_function, num_iterations_seeds);

        calculate_weighted_energies(d_error_weight_0, d_store_energy, d_store_sum_0, d_partition_function, num_iterations_seeds, blocks, j);
        calculate_weighted_energies(d_error_weight_k, d_store_energy, d_store_sum_k, d_partition_function, num_iterations_seeds, blocks, j);

        seeds_interactions += 1;

        //write_bonds(d_interactions, "lattice/final_bonds.txt", nx, ny);
        hipFree(d_store_sum_0);
        hipFree(d_store_sum_k);
        hipFree(d_store_energy);
        hipFree(d_interactions);
        hipFree(d_partition_function);
    }

    float psi = calc_psi(d_error_weight_0, d_error_weight_k, num_iterations_error, nx);

    printf("%f \n", psi/nx); 

    auto t1 = std::chrono::high_resolution_clock::now();
    double duration = (double) std::chrono::duration_cast<std::chrono::seconds>(t1-t0).count();

    printf("Elapsed time for temperature loop sec %f \n", duration);

    hipFree(d_error_weight_0);
    hipFree(d_error_weight_k);
    hipFree(d_wave_vector_k);
}

