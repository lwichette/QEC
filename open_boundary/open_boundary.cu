#include "hip/hip_runtime.h"
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <time.h>
#include <hiprand.h>
#include <hipcub/hipcub.hpp>
#include <thrust/complex.h>
#include <cmath>
#include <math.h>
#include <vector>
#include <string>
#include <sys/stat.h>

#include "../header/defines.h"
#include "../header/utils_big.cuh"

using namespace std;

int main(int argc, char **argv){
    char *results = "results/header";
    int check = create_results_folder(results);
    if (check == 0) return 0;
    
    cout << "Started Simulation" << endl;
    
    // Number iterations and how many lattices
    int num_iterations_seeds = 10;
    int num_iterations_error = 10;
    int niters = 10;
    int nwarmup = 10;
    int num_lattices = 5;

    //prob
    float p = 0.06f;
    
    // Temp
    float start_temp = 1.2f;
    float step = 0.1;

    std::vector<float> inv_temp;
    std::vector<float> coupling_constant;
    float run_temp;

    for (int i=0; i < num_lattices; i++){
        run_temp = start_temp+i*step;
        inv_temp.push_back(1/run_temp);
        coupling_constant.push_back(1/run_temp);
    }

    float *d_inv_temp, *d_coupling_constant;
    hipMalloc(&d_inv_temp, num_lattices*sizeof(float));
    hipMalloc(&d_coupling_constant, num_lattices*sizeof(float));
    hipMemcpy(d_inv_temp, inv_temp.data(), num_lattices*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_coupling_constant, coupling_constant.data(), num_lattices*sizeof(float), hipMemcpyHostToDevice);  

    // Lattice size
    std::array<int, 2> L_size = {28, 36};

    for(int ls = 0; ls < L_size.size(); ls++){
        
        int L = L_size[ls];

        cout << "Started Simulation of Lattice " << L << endl;
        
        // SEEDs
        unsigned long long seeds_spins = 0ULL;
        unsigned long long seeds_interactions = 0ULL;
        
        int blocks = (num_lattices*L*L*2 + THREADS -1)/THREADS;

        auto t0 = std::chrono::high_resolution_clock::now();

        // Allocate the wave vectors and copy it to GPU memory
        std::array<float, 2> wave_vector_0 = {0,0};
        float wv = 2.0f*M_PI/L;
        std::array<float, 2> wave_vector_k = {wv,0};

        float *d_wave_vector_0, *d_wave_vector_k;
        hipMalloc(&d_wave_vector_0, 2 * sizeof(*d_wave_vector_0));
        hipMalloc(&d_wave_vector_k, 2 * sizeof(*d_wave_vector_k));
        hipMemcpy(d_wave_vector_0, wave_vector_0.data(), 2*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_wave_vector_k, wave_vector_k.data(), 2*sizeof(float), hipMemcpyHostToDevice);
        
        //Setup interaction lattice on device
        signed char *d_interactions;
        hipMalloc(&d_interactions, num_lattices*L*L*2*sizeof(*d_interactions));

        // Setup black and white lattice arrays on device
        signed char *lattice_b, *lattice_w;
        hipMalloc(&lattice_b, num_lattices * L * L/2 * sizeof(*lattice_b));
        hipMalloc(&lattice_w, num_lattices * L * L/2 * sizeof(*lattice_w));

        // Weighted error
        float *d_error_weight_0, *d_error_weight_k;
        hipMalloc(&d_error_weight_0, num_lattices*num_iterations_error*sizeof(*d_error_weight_0));
        hipMalloc(&d_error_weight_k, num_lattices*num_iterations_error*sizeof(*d_error_weight_k));

        // Initialize arrays on the GPU to store results per spin system for energy and sum of B2
        thrust::complex<float> *d_store_sum_0, *d_store_sum_k;
        float *d_store_energy;
        hipMalloc(&d_store_sum_0, num_lattices*num_iterations_seeds*sizeof(*d_store_sum_0));
        hipMalloc(&d_store_sum_k, num_lattices*num_iterations_seeds*sizeof(*d_store_sum_k));
        hipMalloc(&d_store_energy, num_lattices*num_iterations_seeds*sizeof(*d_store_energy));

        // B2 Sum 
        thrust::complex<float> *d_sum;
        hipMalloc(&d_sum, num_lattices*L*L/2*sizeof(*d_sum));

        // Weighted energies
        float *d_weighted_energies;
        hipMalloc(&d_weighted_energies, num_lattices*num_iterations_seeds*sizeof(*d_weighted_energies));

        // energy
        float *d_energy;
        hipMalloc(&d_energy, num_lattices*L*L/2*sizeof(*d_energy));

        // Setup cuRAND generator
        hiprandGenerator_t update_rng;
        hiprandCreateGenerator(&update_rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
        float *randvals;
        hipMalloc(&randvals, L * L/2 * sizeof(*randvals));

        // Setup cuRAND generator
        hiprandGenerator_t lattice_rng;
        hiprandCreateGenerator(&lattice_rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
        float *lattice_randvals;
        hipMalloc(&lattice_randvals, num_lattices * L * L/2 * sizeof(*lattice_randvals));

        // Setup cuRAND generator
        hiprandGenerator_t interaction_rng;
        hiprandCreateGenerator(&interaction_rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
        float *interaction_randvals;
        hipMalloc(&interaction_randvals,num_lattices*L*L*2*sizeof(*interaction_randvals));

        // Initialize array for partition function
        float *d_partition_function;
        hipMalloc(&d_partition_function, num_lattices*sizeof(float));

        for (int e = 0; e < num_iterations_error; e++){
            
            cout << "Error " << e << " of " << num_iterations_error << endl;

            init_interactions_with_seed(d_interactions, seeds_interactions, interaction_rng, interaction_randvals, L, L, num_lattices, p);

            for (int s = 0; s < num_iterations_seeds; s++){
                
                init_spins_with_seed(lattice_b, lattice_w, seeds_spins, lattice_rng, lattice_randvals, L, L, num_lattices);

                hiprandSetPseudoRandomGeneratorSeed(update_rng, seeds_spins);
                
                //write_lattice(lattice_b, lattice_w, "lattices/lattice_"+std::to_string(e) + std::string("_") + std::to_string(s) + std::string("_"), L, L, num_lattices);

                hipDeviceSynchronize();

                // Warmup iterations
                //printf("Starting warmup...\n");
                for (int j = 0; j < nwarmup; j++) {
                    update_ob(lattice_b, lattice_w, randvals, update_rng, d_interactions, d_inv_temp, L, L, num_lattices, d_coupling_constant);
                }
                
                hipDeviceSynchronize();

                for (int j = 0; j < niters; j++) {
                    update_ob(lattice_b, lattice_w, randvals, update_rng, d_interactions, d_inv_temp, L, L, num_lattices, d_coupling_constant);
                    //if (j % 1000 == 0) printf("Completed %d/%d iterations...\n", j+1, niters);
                }
                
                hipDeviceSynchronize();

                calculate_B2(d_sum, lattice_b, lattice_w, d_store_sum_0, d_wave_vector_0, s, L, L, num_lattices, num_iterations_seeds);
                calculate_B2(d_sum, lattice_b, lattice_w, d_store_sum_k, d_wave_vector_k, s, L, L, num_lattices, num_iterations_seeds);

                calculate_energy_ob(d_energy, lattice_b, lattice_w, d_interactions, d_store_energy, d_coupling_constant, s, L, L, num_lattices, num_iterations_seeds);

                seeds_spins += 1;
            }

            // Take absolute square + exp
            abs_square<<<blocks, THREADS>>>(d_store_sum_0, num_lattices, num_iterations_seeds);
            abs_square<<<blocks, THREADS>>>(d_store_sum_k, num_lattices, num_iterations_seeds);

            exp_beta<<<blocks, THREADS>>>(d_store_energy, d_inv_temp, num_lattices, num_iterations_seeds, L);
            
            for (int l=0; l<num_lattices; l++){
                hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_store_energy + l*num_iterations_seeds, &d_partition_function[l], num_iterations_seeds);
                hipMalloc(&d_temp, temp_storage);
                hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_store_energy + l*num_iterations_seeds, &d_partition_function[l], num_iterations_seeds);
            }
            
            calculate_weighted_energies(d_weighted_energies, d_error_weight_0, d_store_energy, d_store_sum_0, d_partition_function, num_lattices, num_iterations_seeds, num_iterations_error, blocks, e);
            calculate_weighted_energies(d_weighted_energies, d_error_weight_k, d_store_energy, d_store_sum_k, d_partition_function, num_lattices, num_iterations_seeds, num_iterations_error, blocks, e);

            seeds_interactions += 1;
        }

        // Magnetic susceptibility 
        float *d_magnetic_susceptibility_0, *d_magnetic_susceptibility_k;
        hipMalloc(&d_magnetic_susceptibility_0, num_lattices*sizeof(*d_magnetic_susceptibility_0));
        hipMalloc(&d_magnetic_susceptibility_k, num_lattices*sizeof(*d_magnetic_susceptibility_k));

        for (int l=0; l < num_lattices; l++){
            // Sum reduction for both
            hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_error_weight_0 + l*num_iterations_error, &d_magnetic_susceptibility_0[l], num_iterations_error);
            hipMalloc(&d_temp, temp_storage);
            hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_error_weight_0 + l*num_iterations_error, &d_magnetic_susceptibility_0[l], num_iterations_error);

            hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_error_weight_k + l*num_iterations_error, &d_magnetic_susceptibility_k[l], num_iterations_error);
            hipMalloc(&d_temp, temp_storage);
            hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_error_weight_k + l*num_iterations_error, &d_magnetic_susceptibility_k[l], num_iterations_error);
        }

        hipDeviceSynchronize();

        std::vector<float> h_magnetic_susceptibility_0(num_lattices);
        std::vector<float> h_magnetic_susceptibility_k(num_lattices);
        
        hipMemcpy(h_magnetic_susceptibility_0.data(), d_magnetic_susceptibility_0, num_lattices*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(h_magnetic_susceptibility_k.data(), d_magnetic_susceptibility_k, num_lattices*sizeof(float), hipMemcpyDeviceToHost);

        std::vector<float> psi(num_lattices);
        
        for (int l=0; l < num_lattices; l++){
            psi[l] = (1/(2*sin(M_PI/L))*sqrt(h_magnetic_susceptibility_0[l] / h_magnetic_susceptibility_k[l] - 1))/L;
        }

        auto t1 = std::chrono::high_resolution_clock::now();
        double duration = (double) std::chrono::duration_cast<std::chrono::seconds>(t1-t0).count();

        printf("Elapsed time for temperature loop min %f \n", duration/60);

        // Write results
        std::ofstream f;
        f.open(results + std::string("/L_") + std::to_string(L) + std::string("_ns_") + std::to_string(num_iterations_seeds) + std::string("_ne_") + std::to_string(num_iterations_error) + std::string("_ni_") + std::to_string(niters) + std::string("_nw_") + std::to_string(nwarmup) + std::string(".txt"));
        if (f.is_open()) {
            for (int i = 0; i < num_lattices; i++) {
                f << psi[i] << " " << 1/inv_temp[i] << "\n";
            }
        }
        f.close();

        hipFree(d_wave_vector_0);
        hipFree(d_wave_vector_k);
        hipFree(d_interactions);
        hipFree(lattice_b);
        hipFree(lattice_w);
        hipFree(d_error_weight_0);
        hipFree(d_error_weight_k);
        hipFree(d_store_sum_0);
        hipFree(d_store_sum_k);
        hipFree(d_store_energy);
        hipFree(d_sum);
        hipFree(d_weighted_energies);
        hipFree(d_energy);
        hipFree(randvals);
        hipFree(lattice_randvals);
        hipFree(interaction_randvals);
        hipFree(d_partition_function);
        hipFree(d_magnetic_susceptibility_0);
        hipFree(d_magnetic_susceptibility_k);

        hiprandDestroyGenerator(update_rng);
        hiprandDestroyGenerator(interaction_rng);
        hiprandDestroyGenerator(lattice_rng);
    }
}