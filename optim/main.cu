#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Mauro Bisson <maurob@nvidia.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <getopt.h>
#include <unistd.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "cudamacro.h" /* for time() */
#include "utils.h"
#include <iostream>
#include <thrust/complex.h>
#include <hipcub/hipcub.hpp>
#include <cmath>
#include <ctime>

using namespace std;

#define DIV_UP(a,b)     (((a)+((b)-1))/(b))

#define THREADS  128

// Bits per spin
#define BIT_X_SPIN (4)

// MIN & MAX Operator
#define MIN(a,b)	(((a)<(b))?(a):(b))
#define MAX(a,b)	(((a)>(b))?(a):(b))

// 2048+: 16, 16, 2, 1
//  1024: 16, 16, 1, 2
//   512:  8,  8, 1, 1
//   256:  4,  8, 1, 1
//   128:  2,  8, 1, 1
// 2*SPIN_X_WORD*2*BLOCK_X*BMULT_X
// BMULT_X Block Multiple X Direction

// Unclear
#define BLOCK_X (2)
#define BLOCK_Y (8)

// Unclear
#define BMULT_X (1)
#define BMULT_Y (1)

// Maximum number of GPUs
#define MAX_GPU	(256)

__device__ __forceinline__ unsigned int __mypopc(const unsigned int x) {
	return __popc(x);
}

__device__ __forceinline__ unsigned long long int __mypopc(const unsigned long long int x) {
	return __popcll(x);
}

enum {C_BLACK, C_WHITE};

__device__ __forceinline__ uint2 __mymake_int2(const unsigned int x,
		                               const unsigned int y) {
	return make_uint2(x, y);
}

__device__ __forceinline__ ulonglong2 __mymake_int2(const unsigned long long x,
		                                    const unsigned long long y) {
	return make_ulonglong2(x, y);
}

void *d_temp = NULL;
size_t temp_storage = 0;

void *d_temp_complex = NULL;
size_t temp_storage_complex = 0;

template<int BDIM_X,
	 int BDIM_Y,
	 int LOOP_X,
	 int LOOP_Y,
	 int BITXSP,
	 int COLOR,
	 typename INT_T,
	 typename INT2_T>
__global__  void latticeInit_k(const int devid,
			       const long long seed,
                               const int it,
                               const long long begY,
                               const long long dimX, // ld
                                     INT2_T *__restrict__ vDst) {

	// i linearized y position in blocks and threads
	// j linearized x position in blocks and threads
	const int __i = blockIdx.y*BDIM_Y*LOOP_Y + threadIdx.y;
	const int __j = blockIdx.x*BDIM_X*LOOP_X + threadIdx.x;

	// calculate number of spins per word
	const int SPIN_X_WORD = 8*sizeof(INT_T)/BITXSP;

	// get thread id
	const long long tid = ((devid*gridDim.y + blockIdx.y)*gridDim.x + blockIdx.x)*BDIM_X*BDIM_Y +
	                       threadIdx.y*BDIM_X + threadIdx.x;

	// Random number generator
	hiprandStatePhilox4_32_10_t st;
	// Unclear what exactly
	hiprand_init(seed, tid, static_cast<long long>(2*SPIN_X_WORD)*LOOP_X*LOOP_Y*(2*it+COLOR), &st);

	// tmp 2D array of type unsigned long long of size (1x2)
	INT2_T __tmp[LOOP_Y][LOOP_X];

	// Initialize array with (0,0)
	#pragma unroll //compiler more efficient
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			__tmp[i][j] = __mymake_int2(INT_T(0),INT_T(0));
		}
	}

	//INT = Unsigned long long
	//INT2 == (ull, ull)
	// BIT X SP = 4
	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			#pragma unroll
			for(int k = 0; k < 8*sizeof(INT_T); k += BITXSP) {
				// Logical or plus shifting --> Initialize spins to up or down
				if (hiprand_uniform(&st) < 0.5f) {
					__tmp[i][j].x |= INT_T(1) << k;
				}
				if (hiprand_uniform(&st) < 0.5f) {
					__tmp[i][j].y |= INT_T(1) << k;
				}
			}
		}
	}

	// Set values in overall array
	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			vDst[(begY + __i + i*BDIM_Y)*dimX + __j+j*BDIM_X] = __tmp[i][j];
		}
	}
	return;
}

template<int BDIM_X,
	 int BDIM_Y,
	 int LOOP_X,
	 int LOOP_Y,
	 int BITXSP,
	 typename INT_T,
	 typename INT2_T>
__global__  void hamiltInitB_k(const int devid,
			       const float tgtProb,
			       const long long seed,
                               const long long begY,
                               const long long dimX, // ld
                                     INT2_T *__restrict__ hamB) {

	// i column index in block thread picture, j row index in block thread picture
	const int __i = blockIdx.y*BDIM_Y*LOOP_Y + threadIdx.y;
	const int __j = blockIdx.x*BDIM_X*LOOP_X + threadIdx.x;

	// Thread id
	const long long tid = ((devid*gridDim.y + blockIdx.y)*gridDim.x + blockIdx.x)*BDIM_X*BDIM_Y +
	                       threadIdx.y*BDIM_X + threadIdx.x;

	// Random number generator
	hiprandStatePhilox4_32_10_t st;
	hiprand_init(seed, tid, 0, &st);

	// array of tuples of size (1,2) unsigned long long
	// Set entries to zero tuples of unsigned long long
	INT2_T __tmp[LOOP_Y][LOOP_X];
	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			__tmp[i][j] = __mymake_int2(INT_T(0),INT_T(0));
		}
	}

	// For each black spin, randomly generate 4 interactions
	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			#pragma unroll
			for(int k = 0; k < 8*sizeof(INT_T); k += BITXSP) {
				#pragma unroll
				for(int l = 0; l < BITXSP; l++) {
					if (hiprand_uniform(&st) < tgtProb) {
						__tmp[i][j].x |= INT_T(1) << (k+l);
					}
					if (hiprand_uniform(&st) < tgtProb) {
						__tmp[i][j].y |= INT_T(1) << (k+l);
					}
				}
			}
		}
	}

	// Fill array with the interaction terms
	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			hamB[(begY + __i+i*BDIM_Y)*dimX + __j+j*BDIM_X] = __tmp[i][j];
		}
	}
	return;
}

template<int BDIM_X,
	 int BDIM_Y,
	 int LOOP_X,
	 int LOOP_Y,
	 int BITXSP,
	 typename INT_T,
	 typename INT2_T>
__global__ void hamiltInitW_k(const int xsl,
			      const int ysl,
			      const long long begY,
		              const long long dimX,
		              const INT2_T *__restrict__ hamB,
		                    INT2_T *__restrict__ hamW) {

	// Thread id x and y position
	const int tidx = threadIdx.x;
	const int tidy = threadIdx.y;

	// row and column index of block-thread image
	const int __i = blockIdx.y*BDIM_Y*LOOP_Y + tidy;
	const int __j = blockIdx.x*BDIM_X*LOOP_X + tidx;

	// Array of unsigned long long tuples
	// Load corresponding interactions from hamB
	INT2_T __me[LOOP_Y][LOOP_X];

	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			__me[i][j] = hamB[(begY+__i+i*BDIM_Y)*dimX + __j+j*BDIM_X];
		}
	}

	// Initialize arrays for up/side/down neighbors for white words
	INT2_T __up[LOOP_Y][LOOP_X];
	INT2_T __ct[LOOP_Y][LOOP_X];
	INT2_T __dw[LOOP_Y][LOOP_X];
	INT2_T __sd[LOOP_Y][LOOP_X];

	// the 4 bits of me codify: <upJ, downJ, leftJ, rightJ>
	// 0x888888 --> 100010001000 ...
	// Get first bit in every group of four and shift it by one to the right for up array
	// i.e. get up neighbor of black spin and store it at second position
	// 0x44444 --> 010001000100
	// get second bit in every group of four and shift it by one to the left for down array
	// get down neighbor of black spin and store it at first position
	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			__up[i][j].x = (__me[i][j].x & 0x8888888888888888ull) >> 1;
			__up[i][j].y = (__me[i][j].y & 0x8888888888888888ull) >> 1;

			__dw[i][j].x = (__me[i][j].x & 0x4444444444444444ull) << 1;
			__dw[i][j].y = (__me[i][j].y & 0x4444444444444444ull) << 1;
		}
	}

	// check row parity
	const int readBack = !(__i%2); // this kernel reads only BLACK Js

	// 8*8 = 64 bits per word
	const int BITXWORD = 8*sizeof(INT_T);

	if (!readBack) {
		#pragma unroll
		for(int i = 0; i < LOOP_Y; i++) {
			#pragma unroll
			for(int j = 0; j < LOOP_X; j++) {

				// 0x22222 --> 001000100010
				// get third bit in every group of four and shift it by one to the right for ct array
				// i.e. get leftJ and move it to the fourth position
				__ct[i][j].x = (__me[i][j].x & 0x2222222222222222ull) >> 1;
				__ct[i][j].y = (__me[i][j].y & 0x2222222222222222ull) >> 1;

				// 0x1111 --> 000100010001
				// get fourth bit in every group of four and shift it by (BITXSP + 1) to the left or right ,i.e. to the third position in the
				// prior/next 4 bit group or by (BITXWORD-BITXSP - 1) to the right and perform logical or with already existing ct
				// ct contains then at every 3rd and 4th position in the 4 bits an entry
				__ct[i][j].x |= (__me[i][j].x & 0x1111111111111111ull) << (BITXSP+1);
				__ct[i][j].y |= (__me[i][j].x & 0x1111111111111111ull) >> (BITXWORD-BITXSP - 1);
				__ct[i][j].y |= (__me[i][j].y & 0x1111111111111111ull) << (BITXSP+1);

				// get fourth bit of every four bit group and shift it by 59 to the right -- > Only one entry at the 63th bit
				// set __sd[i][j] = 0
				__sd[i][j].x = (__me[i][j].y & 0x1111111111111111ull) >> (BITXWORD-BITXSP - 1);
				__sd[i][j].y = 0;
			}
		}
	} else {
		#pragma unroll
		for(int i = 0; i < LOOP_Y; i++) {
			#pragma unroll
			for(int j = 0; j < LOOP_X; j++) {
				// Get fourth bit in every group of four and shift it by one to the left, i.e. to the third position
				__ct[i][j].x = (__me[i][j].x & 0x1111111111111111ull) << 1;
				__ct[i][j].y = (__me[i][j].y & 0x1111111111111111ull) << 1;

				// Right part: Get third bit in every group of four and shift it to the fourth position in the next group of four (first and third line)
				// (Second line): Get third bit in every group of four and shift it by 59 to the left --> Only last third bit is at the fourth bit location
				// Logical or: Perform logical or of existing ct with right part to have entries at third and fourth position in each block of four
				__ct[i][j].y |= (__me[i][j].y & 0x2222222222222222ull) >> (BITXSP+1);
				__ct[i][j].x |= (__me[i][j].y & 0x2222222222222222ull) << (BITXWORD-BITXSP - 1);
				__ct[i][j].x |= (__me[i][j].x & 0x2222222222222222ull) >> (BITXSP+1);

				// Get every third bit and shift it by 59 to the left --> last third bit to the fourth position in sd[i][j].y
				__sd[i][j].y = (__me[i][j].x & 0x2222222222222222ull) << (BITXWORD-BITXSP - 1);
				__sd[i][j].x = 0;
			}
		}
	}

	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {

		// calc row index
		const int yoff = begY+__i + i*BDIM_Y;

		// upOff if we are at a boarder of a lattice then take last row, else take row -1
		const int upOff = ( yoff   %ysl) == 0 ? yoff+ysl-1 : yoff-1;
		// downOff: if we are at a lower boarder of a sublattice, take first row, else take row + 1
		const int dwOff = ((yoff+1)%ysl) == 0 ? yoff-ysl+1 : yoff+1;

		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {

			// get column index
			const int xoff = __j + j*BDIM_X;

			// perform logical or with given binary entry
			// yoff*dimX + xoff = pointer at given tuple word lattice entry
			// upOff*dimX + xoff = pointer to upper neighbor
			// dwOff*dimX + xoff = pointer to down neighbor
			// subsequently fills all 64 bits with the up, down, left, right neighbors
			atomicOr(&hamW[yoff*dimX + xoff].x, __ct[i][j].x);
			atomicOr(&hamW[yoff*dimX + xoff].y, __ct[i][j].y);

			atomicOr(&hamW[upOff*dimX + xoff].x, __up[i][j].x);
			atomicOr(&hamW[upOff*dimX + xoff].y, __up[i][j].y);

			atomicOr(&hamW[dwOff*dimX + xoff].x, __dw[i][j].x);
			atomicOr(&hamW[dwOff*dimX + xoff].y, __dw[i][j].y);


			// Depending on which row we are in
			// Check whether we are at bordering columns of sublattices
			// Get column of left right neighbor and perform bitwise or
			const int sideOff = readBack ? (  (xoff   %xsl) == 0 ? xoff+xsl-1 : xoff-1 ):
						       ( ((xoff+1)%xsl) == 0 ? xoff-xsl+1 : xoff+1);

			atomicOr(&hamW[yoff*dimX + sideOff].x, __sd[i][j].x);
			atomicOr(&hamW[yoff*dimX + sideOff].y, __sd[i][j].y);
		}
	}
	return;
}

template<int BDIM_X,
	 int BDIM_Y,
	 int TILE_X,
	 int TILE_Y,
	 int FRAME_X,
	 int FRAME_Y,
	 typename INT2_T>
__device__ void loadTile(const int slX,
			 const int slY,
			 const long long begY,
			 const long long dimX,
			 const INT2_T *__restrict__ v,
			       INT2_T tile[][TILE_X+2*FRAME_X]) {

	// x,y block indices
	const int blkx = blockIdx.x;
	const int blky = blockIdx.y;

	// x,y thread indices
	const int tidx = threadIdx.x;
	const int tidy = threadIdx.y;

	// TILE_X = BLOCK_X*BMULT_X, TILE = [16,32]
	// X and Y startpoint, Y offset by begY depending on GPU
	const int startX =        blkx*TILE_X;
	const int startY = begY + blky*TILE_Y;

	// Loop over BMULT_Y and BMULT_X
	// For each block load Spinwords of size 16x32 in tiles
	#pragma unroll
	for(int j = 0; j < TILE_Y; j += BDIM_Y) {
		// yoffset for current thread idy
		int yoff = startY + j + tidy;

		#pragma unroll
		for(int i = 0; i < TILE_X; i += BDIM_X) {
			// xoffset for current thread idx
			const int xoff = startX + i + tidx;
			tile[FRAME_Y + j + tidy][FRAME_X + i + tidx] = v[yoff*dimX + xoff];
		}
	}

	// if tidy == 0
	if (tidy == 0) {
		// if beginning of Y % size of sublattice == 0 --> if we are at start of a new sublattice
		// set offset to last row, else to startY - 1
		int yoff = (startY % slY) == 0 ? startY+slY-1 : startY-1;

		#pragma unroll
		// Loop over BMULT_Y
		// Get up neighbors
		for(int i = 0; i < TILE_X; i += BDIM_X) {
			const int xoff = startX + i + tidx;
			tile[0][FRAME_X + i + tidx] = v[yoff*dimX + xoff];
		}

		// Down neighbors
		yoff = ((startY+TILE_Y) % slY) == 0 ? startY+TILE_Y - slY : startY+TILE_Y;

		#pragma unroll
		for(int i = 0; i < TILE_X; i += BDIM_X) {
			const int xoff = startX + i+tidx;
			tile[FRAME_Y + TILE_Y][FRAME_X + i + tidx] = v[yoff*dimX + xoff];
		}

		// the other branch in slower so skip it if possible
		// if BLOCK_X <= TILE_Y
		if (BDIM_X <= TILE_Y) {
			// Find left neighbors
			int xoff = (startX % slX) == 0 ? startX+slX-1 : startX-1;

			#pragma unroll
			for(int j = 0; j < TILE_Y; j += BDIM_X) {
				yoff = startY + j+tidx;
				tile[FRAME_Y + j+tidx][0] = v[yoff*dimX + xoff];
			}

			// right neighbors
			xoff = ((startX+TILE_X) % slX) == 0 ? startX+TILE_X - slX : startX+TILE_X;

			#pragma unroll
			for(int j = 0; j < TILE_Y; j += BDIM_X) {
				yoff = startY + j+tidx;
				tile[FRAME_Y + j+tidx][FRAME_X + TILE_X] = v[yoff*dimX + xoff];
			}
		} else {
			// get left and right neighbors
			if (tidx < TILE_Y) {
				int xoff = (startX % slX) == 0 ? startX+slX-1 : startX-1;

				yoff = startY + tidx;
				tile[FRAME_Y + tidx][0] = v[yoff*dimX + xoff];;

				xoff = ((startX+TILE_X) % slX) == 0 ? startX+TILE_X - slX : startX+TILE_X;
				tile[FRAME_Y + tidx][FRAME_X + TILE_X] = v[yoff*dimX + xoff];
			}
		}
	}
	return;
}

template<int BDIM_X,
	 int BDIM_Y,
	 int LOOP_X,
	 int LOOP_Y,
	 int BITXSP,
	 int COLOR,
	 typename INT_T,
	 typename INT2_T>
__global__
void spinUpdate_open_bdry(const int devid,
		      const long long seed,
		      const int it,
		      const int slX, // sublattice size X of one color (in words or word tuples??)
		      const int slY, // sublattice size Y
		      const long long begY,
		      const long long dimX, // ld
		      const float vExp[][5],
		      const INT2_T *__restrict__ jDst,
		      const INT2_T *__restrict__ vSrc,
		            INT2_T *__restrict__ vDst) {

	// calc how many spins per word
	const int SPIN_X_WORD = 8*sizeof(INT_T)/BITXSP;

	// x and y location in Thread lattice
	const int tidx = threadIdx.x;
	const int tidy = threadIdx.y;

	// Initialize shared memory of Block size + neighbors
	__shared__ INT2_T shTile[BDIM_Y*LOOP_Y+2][BDIM_X*LOOP_X+2];

	// Load spin tiles of opposite lattice
	loadTile<BDIM_X, BDIM_Y,
		 BDIM_X*LOOP_X,
		 BDIM_Y*LOOP_Y,
		 1, 1, INT2_T>(slX, slY, begY, dimX, vSrc, shTile);

	// __shExp[cur_s{0,1}][sum_s{0,1}] = __expf(-2*cur_s{-1,+1}*F{+1,-1}(sum_s{0,1})*INV_TEMP)
	// Shared memory to store Exp
	__shared__ float __shExp[2][5];

	// for small lattices BDIM_X/Y may be smaller than 2/5
	// Load exponentials into shared memory
	#pragma unroll
	for(int i = 0; i < 2; i += BDIM_Y) {
		#pragma unroll
		for(int j = 0; j < 5; j += BDIM_X) {
			if (i+tidy < 2 && j+tidx < 5) {
				__shExp[i+tidy][j+tidx] = vExp[i+tidy][j+tidx];
			}
		}
	}
	__syncthreads();

	// get i and j location in block/thread grid
	const int __i = blockIdx.y*BDIM_Y*LOOP_Y + tidy;
	const int __j = blockIdx.x*BDIM_X*LOOP_X + tidx;

	// calculate thread id
	const long long tid = ((devid*gridDim.y + blockIdx.y)*gridDim.x + blockIdx.x)*BDIM_X*BDIM_Y +
	                       threadIdx.y*BDIM_X + threadIdx.x;

	// array of size BMULT_Y x BMULT_X of unsigned long long
	INT2_T __me[LOOP_Y][LOOP_X];

	// Store spin words in array
	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			__me[i][j] = vDst[(begY+__i+i*BDIM_Y)*dimX + __j+j*BDIM_X];
		}
	}

	// initialize up, down center arrays
	INT2_T __up[LOOP_Y][LOOP_X];
	INT2_T __ct[LOOP_Y][LOOP_X];
	INT2_T __dw[LOOP_Y][LOOP_X];

	// Load up, down, center neighbors from other word lattice
	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			// up has +0 in y direction index as shift by additional row in load tile. Same row plus one accordingly and only one down a plus two.
			// same x direction thread goes to plus one by additional entry in x direction in loadTile, too.
			__up[i][j] = shTile[i*BDIM_Y +   tidy][j*BDIM_X + 1+tidx];
			__ct[i][j] = shTile[i*BDIM_Y + 1+tidy][j*BDIM_X + 1+tidx];
			__dw[i][j] = shTile[i*BDIM_Y + 2+tidy][j*BDIM_X + 1+tidx];
		}
	}

	// BDIM_Y is power of two so row parity won't change across loops
	// Check which color and whether row (__i) is even or odd
	// Example: black lattice, even row --> readBack = 1
	const int readBack = (COLOR == C_BLACK) ? !(__i%2) : (__i%2);

	// Load missing side neighbors
	INT2_T __sd[LOOP_Y][LOOP_X];

	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			// Hence with read back we are missing left neighbor and without readback missing right neighbor in center tile
			__sd[i][j] = (readBack) ? shTile[i*BDIM_Y + 1+tidy][j*BDIM_X +   tidx]:
						  shTile[i*BDIM_Y + 1+tidy][j*BDIM_X + 2+tidx];
		}
	}

	// if read back true: Left neighbor of most left spin entry in me must be deduced from rightest spin in sd array and combined with remaining spins from ct.
	// if read back false: right neighbor of most right spin entry in me must be deduced from leftest spin in sd array and combined with remaining spins from ct.

	// Where we ended
	// Rearrange left and right neighbors and update __sd[i,j] by filling it with the "right" neighbors
	// which become left neighbors
	if (readBack) {
		#pragma unroll
		// (BLACK LATTICE) Shift __sd such that it contains the left neighbors of the corresponding __me word
		// (BLACK LATTICE) __ct then contains the right neighbors
		for(int i = 0; i < LOOP_Y; i++) {
			#pragma unroll
			for(int j = 0; j < LOOP_X; j++) {
				__sd[i][j].x = (__ct[i][j].x << BITXSP) | (__sd[i][j].y >> (8*sizeof(__sd[i][j].y)-BITXSP)); // looks like furthest spin on the left side is in binary rep most at most right position!
				__sd[i][j].y = (__ct[i][j].y << BITXSP) | (__ct[i][j].x >> (8*sizeof(__ct[i][j].x)-BITXSP)); // only the x word needs left neighbor from the sd array. the y word gets its remaining spin from the x word.
			}
		}
	} else {
		// (BLACK LATTICE) Shift __sd such that it contains the right neighbors of the corresponding __me word
		// (BLACK LATTICE) __ct then contains the left neighbors
		#pragma unroll
		for(int i = 0; i < LOOP_Y; i++) {
			#pragma unroll
			for(int j = 0; j < LOOP_X; j++) {
				__sd[i][j].y = (__ct[i][j].y >> BITXSP) | (__sd[i][j].x << (8*sizeof(__sd[i][j].x)-BITXSP));
				__sd[i][j].x = (__ct[i][j].x >> BITXSP) | (__ct[i][j].y << (8*sizeof(__ct[i][j].y)-BITXSP));
			}
		}
	}

	// When Hamiltonian is used
	if (jDst != NULL) {
		// Initialize array of size (1,2) to store the interaction terms
		INT2_T __J[LOOP_Y][LOOP_X];

		// Load interactions for current word tuple we are in
		#pragma unroll
		for(int i = 0; i < LOOP_Y; i++) {
			#pragma unroll
			for(int j = 0; j < LOOP_X; j++) {
				__J[i][j] = jDst[(begY+__i+i*BDIM_Y)*dimX + __j+j*BDIM_X];
			}
		}


		// Now the idea is to apply and operation with 0x088088..ull with 0 at positions of open boundary in correct direction and correct spin position in the word
		// after the XOR and the shift the result looks like 0000|0001|0001|.. for each direction the results are than summed.
		// Hence, by setting the 4 bit group to 0000 would result in not regarding this hamiltonian term.
		// to set a 4 bit group to zero it is sufficient to execute a bitwise & with 0x088088..ull with 0 at the place of spins at the boundary
		// There can be the case of all spins inside a word at the boundary 0x0000..ull or only one spin in direction left or right 0x0888..ull and 0x8888..0ull.
		// The left right direction choice is dependent on the color and row parity.

		// apply them
		// the 4 bits of J codify: <upJ, downJ, leftJ, rightJ>
		#pragma unroll
		for(int i = 0; i < LOOP_Y; i++) {
			#pragma unroll
			for(int j = 0; j < LOOP_X; j++) {

				// Perform bitwise or operation
				// Column of left side gets the first bit in every group of four which is then shifted by 3 to the right left because spins are only
				// at the fourth location
				// XOR is then performed to change sign of spins

				__up[i][j].x ^= (__J[i][j].x & 0x8888888888888888ull) >> 3;
				__up[i][j].y ^= (__J[i][j].y & 0x8888888888888888ull) >> 3;

				// get down interaction and shift it to the right place
				__dw[i][j].x ^= (__J[i][j].x & 0x4444444444444444ull) >> 2;
				__dw[i][j].y ^= (__J[i][j].y & 0x4444444444444444ull) >> 2;

				if (readBack) {
					// __sd[][] holds "left" spins
					// __ct[][] holds "right" spins
					// get left interaction and shift it to the right position
					__sd[i][j].x ^= (__J[i][j].x & 0x2222222222222222ull) >> 1; // the shift is executed before the or operation!
					__sd[i][j].y ^= (__J[i][j].y & 0x2222222222222222ull) >> 1;

					// get right interaction and shift it to the right position
					__ct[i][j].x ^= (__J[i][j].x & 0x1111111111111111ull);
					__ct[i][j].y ^= (__J[i][j].y & 0x1111111111111111ull);


					// if me word at left boundary of sublattice - only j=0 should give true here but I still have to check for it's value.
					// only if black and even row possible left neighbor spin is boundary or if white and odd row left neighbor is boundary
					if((__j+j*BDIM_X)%(slX)==0){
						__sd[i][j].x &= 0x1111111111111110ull; // maps most right spin values in array to zero which should be interaction term for most left spin in lattice
					}

				} else {
					// __ct[][] holds "left" spins
					// __sd[][] holds "right" spins
					// get left interaction and shift it to the right position and perform XOR
					__ct[i][j].x ^= (__J[i][j].x & 0x2222222222222222ull) >> 1;
					__ct[i][j].y ^= (__J[i][j].y & 0x2222222222222222ull) >> 1;

					// get right interaction and perform XOR
					__sd[i][j].x ^= (__J[i][j].x & 0x1111111111111111ull);
					__sd[i][j].y ^= (__J[i][j].y & 0x1111111111111111ull);

					// only if black and odd row or white even row the right neighbor may be a sublattice boundary
					if((__j+j*BDIM_X+1)%(slX)==0){
						__sd[i][j].y &= 0x0111111111111111ull; // maps most left spin values in array to zero which should be interaction term for most right spin in lattice
					}
				}
			}
		}
	}

	hiprandStatePhilox4_32_10_t st;
	hiprand_init(seed, tid, static_cast<long long>(2*SPIN_X_WORD)*LOOP_X*LOOP_Y*(2*it+COLOR), &st);

	// Add binaries up but why though
	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			// Check whether current row index on grid is multiple of sublattice dimension y. In this case the investigated word spins lay all at an upper edge of a sublattice.
			// Hence, only the dw, sd, ct boundaries shall be included in this case.
			if((__i+i*BDIM_Y)%slY==0){
					__dw[i][j].x += __sd[i][j].x;
					__ct[i][j].x += __dw[i][j].x;

					__dw[i][j].y += __sd[i][j].y;
					__ct[i][j].y += __dw[i][j].y;
			}
			// Check whether current row index on grid plus one is multiple of sublattice dimension y. In this case the investigated word spins lay all at an lower edge of a sublattice.
			// Hence, only the up, sd, ct boundaries shall be included in this case.
			else if((__i+i*BDIM_Y+1)%slY==0){
					__ct[i][j].x += __up[i][j].x;
					__ct[i][j].x += __sd[i][j].x;

					__ct[i][j].y += __up[i][j].y;
					__ct[i][j].y += __sd[i][j].y;
			}
			// For left an right boundaries was taken care of a step beforehand and thus one can sum over all neighbors here altough the word spins may include left or right boundaries of a sublattice.
			else{
				__ct[i][j].x += __up[i][j].x;
				__dw[i][j].x += __sd[i][j].x;
				__ct[i][j].x += __dw[i][j].x;

				__ct[i][j].y += __up[i][j].y;
				__dw[i][j].y += __sd[i][j].y;
				__ct[i][j].y += __dw[i][j].y;
			}
		}
	}

	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			#pragma unroll
			for(int z = 0; z < 8*sizeof(INT_T); z += BITXSP) {

				// do we need other exponential or random uniform distribution for boundary terms to be correct???

				//__src tuple, perform bitwise operation with 4 bits of __me and 1111
				// Extract information whether spin is up or down --> results in 0 or 1
				const int2 __src = make_int2((__me[i][j].x >> z) & 0xF,
							     (__me[i][j].y >> z) & 0xF);

				// __sum tuple, perform bitwise operation with 4 bits of __ct and 1111
				// Get number of up neighbors for each spin contained in the words --> results in range zero to 4
				const int2 __sum = make_int2((__ct[i][j].x >> z) & 0xF,
							     (__ct[i][j].y >> z) & 0xF);

				// Create unsigned long long 1
				const INT_T ONE = static_cast<INT_T>(1);

				// perform logical XOR on the bits containing the spins
				// updates the spins from -1 to 1 or vice versa
				if (hiprand_uniform(&st) <= __shExp[__src.x][__sum.x]) {
					__me[i][j].x ^= ONE << z;
				}
				if (hiprand_uniform(&st) <= __shExp[__src.y][__sum.y]) {
					__me[i][j].y ^= ONE << z;
				}
			}
		}
	}

	// Store updated spins in the lattice
	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			vDst[(begY + __i+i*BDIM_Y)*dimX + __j+j*BDIM_X] = __me[i][j];
		}
	}
	return;
}

template<int BDIM_X,
	 int BDIM_Y,
	 int LOOP_X,
	 int LOOP_Y,
	 int BITXSP,
	 int COLOR,
	 typename INT_T,
	 typename INT2_T>
__global__
void spinUpdateV_2D_k(const int devid,
		      const long long seed,
		      const int it,
		      const int slX, // sublattice size X of one color (in words)
		      const int slY, // sublattice size Y of one color
		      const long long begY,
		      const long long dimX, // ld
		      const float vExp[][5],
		      const INT2_T *__restrict__ jDst,
		      const INT2_T *__restrict__ vSrc,
		            INT2_T *__restrict__ vDst) {

	// calc how many spins per word
	const int SPIN_X_WORD = 8*sizeof(INT_T)/BITXSP;

	// x and y location in Thread lattice
	const int tidx = threadIdx.x;
	const int tidy = threadIdx.y;

	// Initialize shared memory of Block size + neighbors
	__shared__ INT2_T shTile[BDIM_Y*LOOP_Y+2][BDIM_X*LOOP_X+2];

	// Load spin tiles of opposite lattice
	loadTile<BDIM_X, BDIM_Y,
		 BDIM_X*LOOP_X,
		 BDIM_Y*LOOP_Y,
		 1, 1, INT2_T>(slX, slY, begY, dimX, vSrc, shTile);

	// __shExp[cur_s{0,1}][sum_s{0,1}] = __expf(-2*cur_s{-1,+1}*F{+1,-1}(sum_s{0,1})*INV_TEMP)
	// Shared memory to store Exp
	__shared__ float __shExp[2][5];

	// for small lattices BDIM_X/Y may be smaller than 2/5
	// Load exponentials into shared memory
	#pragma unroll
	for(int i = 0; i < 2; i += BDIM_Y) {
		#pragma unroll
		for(int j = 0; j < 5; j += BDIM_X) {
			if (i+tidy < 2 && j+tidx < 5) {
				__shExp[i+tidy][j+tidx] = vExp[i+tidy][j+tidx];
			}
		}
	}
	__syncthreads();

	// get i and j location in block/thread grid
	const int __i = blockIdx.y*BDIM_Y*LOOP_Y + tidy;
	const int __j = blockIdx.x*BDIM_X*LOOP_X + tidx;

	// calculate thread id
	const long long tid = ((devid*gridDim.y + blockIdx.y)*gridDim.x + blockIdx.x)*BDIM_X*BDIM_Y +
	                       threadIdx.y*BDIM_X + threadIdx.x;

	// array of size BMULT_Y x BMULT_X of unsigned long long
	INT2_T __me[LOOP_Y][LOOP_X];

	// Store spin words in array
	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			__me[i][j] = vDst[(begY+__i+i*BDIM_Y)*dimX + __j+j*BDIM_X];
		}
	}

	// initialize up, down center arrays
	INT2_T __up[LOOP_Y][LOOP_X];
	INT2_T __ct[LOOP_Y][LOOP_X];
	INT2_T __dw[LOOP_Y][LOOP_X];

	// Load up, down, center neighbors from other word lattice
	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			__up[i][j] = shTile[i*BDIM_Y +   tidy][j*BDIM_X + 1+tidx];
			__ct[i][j] = shTile[i*BDIM_Y + 1+tidy][j*BDIM_X + 1+tidx];
			__dw[i][j] = shTile[i*BDIM_Y + 2+tidy][j*BDIM_X + 1+tidx];
		}
	}

	// BDIM_Y is power of two so row parity won't change across loops
	// Check which color and whether row (__i) is even or odd
	// Example: black lattice, even row --> readBack = 1
	const int readBack = (COLOR == C_BLACK) ? !(__i%2) : (__i%2);

	// Load missing side neighbors
	INT2_T __sd[LOOP_Y][LOOP_X];

	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			__sd[i][j] = (readBack) ? shTile[i*BDIM_Y + 1+tidy][j*BDIM_X +   tidx]:
						  shTile[i*BDIM_Y + 1+tidy][j*BDIM_X + 2+tidx];
		}
	}

	// Rearrange left and right neighbors and update __sd[i,j] by filling it with the "right" neighbors
	// which become left neighbors
	if (readBack) {
		#pragma unroll
		// (BLACK LATTICE) Shift __sd such that it contains the left neighbors of the corresponding __me word
		// (BLACK LATTICE) __ct then contains the right neighbors
		for(int i = 0; i < LOOP_Y; i++) {
			#pragma unroll
			for(int j = 0; j < LOOP_X; j++) {
				__sd[i][j].x = (__ct[i][j].x << BITXSP) | (__sd[i][j].y >> (8*sizeof(__sd[i][j].y)-BITXSP));
				__sd[i][j].y = (__ct[i][j].y << BITXSP) | (__ct[i][j].x >> (8*sizeof(__ct[i][j].x)-BITXSP));
			}
		}
	} else {
		// (BLACK LATTICE) Shift __sd such that it contains the right neighbors of the corresponding __me word
		// (BLACK LATTICE) __ct then contains the left neighbors
		#pragma unroll
		for(int i = 0; i < LOOP_Y; i++) {
			#pragma unroll
			for(int j = 0; j < LOOP_X; j++) {
				__sd[i][j].y = (__ct[i][j].y >> BITXSP) | (__sd[i][j].x << (8*sizeof(__sd[i][j].x)-BITXSP));
				__sd[i][j].x = (__ct[i][j].x >> BITXSP) | (__ct[i][j].y << (8*sizeof(__ct[i][j].y)-BITXSP));
			}
		}
	}

	// When Hamiltonian is used
	if (jDst != NULL) {
		// Initialize array of size (1,2) to store the interaction terms
		INT2_T __J[LOOP_Y][LOOP_X];

		// Load interactions for current word tuple we are in
		#pragma unroll
		for(int i = 0; i < LOOP_Y; i++) {
			#pragma unroll
			for(int j = 0; j < LOOP_X; j++) {
				__J[i][j] = jDst[(begY+__i+i*BDIM_Y)*dimX + __j+j*BDIM_X];
			}
		}

		// apply them
		// the 4 bits of J codify: <upJ, downJ, leftJ, rightJ>
		#pragma unroll
		for(int i = 0; i < LOOP_Y; i++) {
			#pragma unroll
			for(int j = 0; j < LOOP_X; j++) {

				// Perform bitwise or operation
				// Column of left side gets the first bit in every group of four which is then shifted by 3 to the right left because spins are only
				// at the fourth location
				// XOR is then performed to change sign of spins
				__up[i][j].x ^= (__J[i][j].x & 0x8888888888888888ull) >> 3;
				__up[i][j].y ^= (__J[i][j].y & 0x8888888888888888ull) >> 3;

				// get down interaction and shift it to the right place
				__dw[i][j].x ^= (__J[i][j].x & 0x4444444444444444ull) >> 2;
				__dw[i][j].y ^= (__J[i][j].y & 0x4444444444444444ull) >> 2;

				if (readBack) {
					// __sd[][] holds "left" spins
					// __ct[][] holds "right" spins
					// get left interaction and shift it to the right position
					__sd[i][j].x ^= (__J[i][j].x & 0x2222222222222222ull) >> 1;
					__sd[i][j].y ^= (__J[i][j].y & 0x2222222222222222ull) >> 1;

					// get right interaction and shift it to the right position
					__ct[i][j].x ^= (__J[i][j].x & 0x1111111111111111ull);
					__ct[i][j].y ^= (__J[i][j].y & 0x1111111111111111ull);
				} else {
					// __ct[][] holds "left" spins
					// __sd[][] holds "right" spins
					// get left interaction and shift it to the right position and perform XOR
					__ct[i][j].x ^= (__J[i][j].x & 0x2222222222222222ull) >> 1;
					__ct[i][j].y ^= (__J[i][j].y & 0x2222222222222222ull) >> 1;

					// get right interaction and perform XOR
					__sd[i][j].x ^= (__J[i][j].x & 0x1111111111111111ull);
					__sd[i][j].y ^= (__J[i][j].y & 0x1111111111111111ull);
				}
			}
		}
	}

	hiprandStatePhilox4_32_10_t st;
	hiprand_init(seed, tid, static_cast<long long>(2*SPIN_X_WORD)*LOOP_X*LOOP_Y*(2*it+COLOR), &st);

	// Add binaries up but why though
	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			// __ct contains at the end number of neighboring up spins in binary for the two words x and y
			__ct[i][j].x += __up[i][j].x;
			__dw[i][j].x += __sd[i][j].x;
			__ct[i][j].x += __dw[i][j].x;

			__ct[i][j].y += __up[i][j].y;
			__dw[i][j].y += __sd[i][j].y;
			__ct[i][j].y += __dw[i][j].y;
		}
	}

	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			#pragma unroll
			for(int z = 0; z < 8*sizeof(INT_T); z += BITXSP) {

				//__src tuple, perform bitwise operation with 4 bits of __me and 1111
				// Extract information whether spin is up or down --> results in 0 or 1
				const int2 __src = make_int2((__me[i][j].x >> z) & 0xF,
							     (__me[i][j].y >> z) & 0xF);

				// __sum tuple, perform bitwise operation with 4 bits of __ct and 1111
				// Get number of up neighbors for each spin contained in the words --> results in range zero to 4
				const int2 __sum = make_int2((__ct[i][j].x >> z) & 0xF,
							     (__ct[i][j].y >> z) & 0xF);

				// Create unsigned long long 1
				const INT_T ONE = static_cast<INT_T>(1);

				// perform logical XOR on the bits containing the spins
				// updates the spins from -1 to 1 or vice versa
				if (hiprand_uniform(&st) <= __shExp[__src.x][__sum.x]) {
					__me[i][j].x ^= ONE << z;
				}
				if (hiprand_uniform(&st) <= __shExp[__src.y][__sum.y]) {
					__me[i][j].y ^= ONE << z;
				}
			}
		}
	}

	// Store updated spins in the lattice
	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			vDst[(begY + __i+i*BDIM_Y)*dimX + __j+j*BDIM_X] = __me[i][j];
		}
	}
	return;
}

template<int BDIM_X,
	 int WSIZE,
	 typename T>
__device__ __forceinline__ T __block_sum(T v) {

	__shared__ T sh[BDIM_X/WSIZE];

	const int lid = threadIdx.x%WSIZE;
	const int wid = threadIdx.x/WSIZE;

	#pragma unroll
	for(int i = WSIZE/2; i; i >>= 1) {
		v += __shfl_down_sync(0xFFFFFFFF, v, i);
	}
	if (lid == 0) sh[wid] = v;

	__syncthreads();
	if (wid == 0) {
		v = (lid < (BDIM_X/WSIZE)) ? sh[lid] : 0;

		#pragma unroll
		for(int i = (BDIM_X/WSIZE)/2; i; i >>= 1) {
			v += __shfl_down_sync(0xFFFFFFFF, v, i);
		}
	}
	__syncthreads();
	return v;
}

// to be optimized
template<int BDIM_X,
	 int BITXSP,
         typename INT_T,
	 typename SUM_T>
__global__ void getMagn_k(const long long n, // llen
			  const INT_T *__restrict__ v, // black_d
			        SUM_T *__restrict__ sum) {

	// Get number of spins per word
	const int SPIN_X_WORD = 8*sizeof(INT_T)/BITXSP;

	// nth = blockDim.x*gridDim.x???
	const long long nth = static_cast<long long>(blockDim.x)*gridDim.x;
	const long long tid = static_cast<long long>(blockDim.x)*blockIdx.x + threadIdx.x;

	// counter for positive and negative
	SUM_T __cntP = 0;
	SUM_T __cntN = 0;

	// Loop over all entries until end of array is found
	for(long long i = 0; i < n; i += nth) {
		// Check if still in range
		if (i+tid < n) {
			// counts the number of non_zero bits in v[i+tid]
			// Add up correspondingly
			const int __c = __mypopc(v[i+tid]);
			__cntP += __c;
			__cntN += SPIN_X_WORD - __c;
		}
	}

	__cntP = __block_sum<BDIM_X, 32>(__cntP);
	__cntN = __block_sum<BDIM_X, 32>(__cntN);

	if (threadIdx.x == 0) {
		atomicAdd(sum+0, __cntP);
		atomicAdd(sum+1, __cntN);
	}
	return;
}

static void countSpins(const int ndev,
		       const int redBlocks,
		       const size_t llen,
		       const size_t llenLoc,
		       const unsigned long long *black_d,
		       const unsigned long long *white_d,
			     unsigned long long **sum_d,
			     unsigned long long *bsum,
			     unsigned long long *wsum) {

	if (ndev == 1) {
		CHECK_CUDA(hipMemset(sum_d[0], 0, 2*sizeof(**sum_d)));
		getMagn_k<THREADS, BIT_X_SPIN><<<redBlocks, THREADS>>>(llen, black_d, sum_d[0]);
		CHECK_ERROR("getMagn_k");
		CHECK_CUDA(hipDeviceSynchronize());
	}

	else {
		for(int i = 0; i < ndev; i++) {

			CHECK_CUDA(hipSetDevice(i));
			CHECK_CUDA(hipMemset(sum_d[i], 0, 2*sizeof(**sum_d)));
			getMagn_k<THREADS, BIT_X_SPIN><<<redBlocks, THREADS>>>(llenLoc, black_d + i*llenLoc, sum_d[i]);
			getMagn_k<THREADS, BIT_X_SPIN><<<redBlocks, THREADS>>>(llenLoc, white_d + i*llenLoc, sum_d[i]);
			CHECK_ERROR("getMagn_k");
		}
	}

	bsum[0] = 0;
	wsum[0] = 0;

	unsigned long long  sum_h[MAX_GPU][2];

	for(int i = 0; i < ndev; i++) {
		CHECK_CUDA(hipMemcpy(sum_h[i], sum_d[i], 2*sizeof(**sum_h), hipMemcpyDeviceToHost));
		bsum[0] += sum_h[i][0];
		wsum[0] += sum_h[i][1];
	}
	return;
}

static void dumpLattice(const char *fprefix,
			const int ndev,
			const int Y,
			const size_t lld,
		        const size_t llen,
		        const size_t llenLoc,
		        const unsigned long long *v_d) {

	char fname[256];

	if (ndev == 1) {
		unsigned long long *v_h = (unsigned long long *)Malloc(llen*sizeof(*v_h));
		CHECK_CUDA(hipMemcpy(v_h, v_d, llen*sizeof(*v_h), hipMemcpyDeviceToHost));

		unsigned long long *black_h = v_h;
		unsigned long long *white_h = v_h + llen/2;

		snprintf(fname, sizeof(fname), "%s0.txt", fprefix);
		FILE *fp = Fopen(fname, "w");

		for(int i = 0; i < Y; i++) {
			for(int j = 0; j < lld; j++) {
				unsigned long long __b = black_h[i*lld + j];
				unsigned long long __w = white_h[i*lld + j];

				for(int k = 0; k < 8*sizeof(*v_h); k += BIT_X_SPIN) {
					if (i&1) {
						fprintf(fp, "%llX",  (__w >> k) & 0xF);
						fprintf(fp, "%llX",  (__b >> k) & 0xF);
					} else {
						fprintf(fp, "%llX",  (__b >> k) & 0xF);
						fprintf(fp, "%llX",  (__w >> k) & 0xF);
					}
				}
			}
			fprintf(fp, "\n");
		}
		fclose(fp);
		free(v_h);
	} else {
		#pragma omp parallel for schedule(static)
		for(int d = 0; d < ndev; d++) {
			const unsigned long long *black_h = v_d +          d*llenLoc;
			const unsigned long long *white_h = v_d + llen/2 + d*llenLoc;

			snprintf(fname, sizeof(fname), "%s%d.txt", fprefix, d);
			FILE *fp = Fopen(fname, "w");

			for(int i = 0; i < Y; i++) {
				for(int j = 0; j < lld; j++) {
					unsigned long long __b = black_h[i*lld + j];
					unsigned long long __w = white_h[i*lld + j];

					for(int k = 0; k < 8*sizeof(*black_h); k += BIT_X_SPIN) {
						if (i&1) {
							fprintf(fp, "%llX",  (__w >> k) & 0xF);
							fprintf(fp, "%llX",  (__b >> k) & 0xF);
						} else {
							fprintf(fp, "%llX",  (__b >> k) & 0xF);
							fprintf(fp, "%llX",  (__w >> k) & 0xF);
						}
					}
				}
				fprintf(fp, "\n");
			}
			fclose(fp);
		}
	}
	return;
}

template<int BDIM_X,
	 int BDIM_Y,
	 int LOOP_X,
	 int LOOP_Y,
	 int BITXSP,
	 typename INT_T,
	 typename INT2_T>
__global__ void calculate_average_magnetization(const int devid,
			const int slX,
			const int slY,
			const long long begY,
			const long long dimX,
			const INT2_T *__restrict__ v_white,
			const INT2_T *__restrict__ v_black,
			const thrust::complex<float> *exp,
			const int blocks_per_slx,
			const int blocks_per_sly,
			int *sum_per_block,
			thrust::complex<float> *c_sum_per_block) {

	// calc how many spins per word
	const int SPIN_X_WORD = 8*sizeof(INT_T)/BITXSP;

	// x and y location in Thread lattice
	const int tidx = threadIdx.x;
	const int tidy = threadIdx.y;

	const int __i = blockIdx.y*BDIM_Y*LOOP_Y + tidy;

	// Initialize shared memory of Block size + neighbors
	__shared__ INT2_T shTile_w[BDIM_Y*LOOP_Y+2][BDIM_X*LOOP_X+2];
	__shared__ INT2_T shTile_b[BDIM_Y*LOOP_Y+2][BDIM_X*LOOP_X+2];

	// Store sum
	__shared__ int sum[BDIM_Y*BDIM_X];
	__shared__ thrust::complex<float> c_sum[BDIM_Y*BDIM_X];

	// Load spin tiles of lattice
	loadTile<BDIM_X, BDIM_Y,
		 BDIM_X*LOOP_X,
		 BDIM_Y*LOOP_Y,
		 1, 1, INT2_T>(slX, slY, begY, dimX, v_white, shTile_w);

	__syncthreads();

	loadTile<BDIM_X, BDIM_Y,
		 BDIM_X*LOOP_X,
		 BDIM_Y*LOOP_Y,
		 1, 1, INT2_T>(slX, slY, begY, dimX, v_black, shTile_b);

	__syncthreads();

	// array of size BMULT_Y x BMULT_X of unsigned long long
	INT2_T __me_w[LOOP_Y][LOOP_X];
	INT2_T __me_b[LOOP_Y][LOOP_X];

	// Store spin words in array
	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			__me_w[i][j] = shTile_w[1 + tidy + i*BDIM_Y][1 + tidx + j*BDIM_X];
			__me_b[i][j] = shTile_b[1 + tidy + i*BDIM_Y][1 + tidx + j*BDIM_X];
		}
	}

	int __cntP = 0;
	int __cntN = 0;

	thrust::complex<float> run_sum = thrust::complex<float>(0.0, 0.0);

	// Store spin words in array
	#pragma unroll
	for(int i = 0; i < LOOP_Y; i++) {
		const int __sli = (__i+i*BDIM_Y) % slY;
		#pragma unroll
		for(int j = 0; j < LOOP_X; j++) {
			const int __cx = __popcll(__me_w[i][j].x) + __popcll(__me_b[i][j].x);
			const int __cy = __popcll(__me_w[i][j].y) + __popcll(__me_b[i][j].y);

			__cntP += (__cx + __cy);
			__cntN += 4*SPIN_X_WORD - __cx - __cy;

			int spin_sum = __cx + __cy - (4*SPIN_X_WORD - __cx - __cy);
			run_sum += exp[__sli]*spin_sum;
			//run_sum = hipCaddf(hipCmulf(exp[__sli], make_hipFloatComplex(spin_sum, 0)), run_sum);
		}
	}

	sum[tidy*BDIM_X + tidx] = __cntP - __cntN;
	c_sum[tidy*BDIM_X +tidx] = run_sum;

	__syncthreads();

	for (int s = blockDim.y*blockDim.x/2; s>0; s >>= 1){
		if (tidy*BDIM_X + tidx < s){
			sum[tidy*BDIM_X + tidx] += sum[tidy*BDIM_X + tidx + s];
			c_sum[tidy*BDIM_X + tidx] += c_sum[tidy*BDIM_X + tidx + s];
		}
		__syncthreads();
	}

	if ((tidx == 0) & (tidy == 0)){
		const int current_x = blockIdx.x/blocks_per_slx;
		const int current_y = blockIdx.y/blocks_per_sly;
		const int offset = (current_y*gridDim.x/blocks_per_slx + current_x)*blocks_per_slx*blocks_per_sly;

		const int block_lin_y = blockIdx.y%blocks_per_sly;
		const int block_lin_x = blockIdx.x%blocks_per_slx;

		sum_per_block[offset + block_lin_y*blocks_per_slx + block_lin_x] = sum[0];
		c_sum_per_block[offset + block_lin_y*blocks_per_slx + block_lin_x] = c_sum[0];
	}
}

__global__ void calculate_incremental_susceptibility(const int blocks_per_slx,
				const int blocks_per_sly,
				const int num_lattices,
				const int *d_sums_per_block,
				const thrust::complex<float> *d_weighted_sums_per_block,
				int *d_store_sum,
				float *d_sus_k){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = tid*blocks_per_slx*blocks_per_sly;

	if (tid >= num_lattices) return;

	int sum = 0;
	thrust::complex<float> c_sum = thrust::complex<float> (0.0f, 0.0f);

	for (int i=0; i < blocks_per_slx*blocks_per_sly; i++){
		sum += d_sums_per_block[offset + i];
		c_sum += d_weighted_sums_per_block[offset + i];
	}

	d_store_sum[tid] += pow(abs(sum), 2);
	d_sus_k[tid] += thrust::abs(c_sum)*thrust::abs(c_sum);
}


__global__ void incremental_susceptibility(int *d_store_sum,
			thrust::complex<float> *d_store_weighted_sum,
			int *d_inc_sus_0,
			float *d_inc_sus_k,
			int num_lattices){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid >= num_lattices) return;

	d_inc_sus_0[tid] += pow(abs(d_store_sum[tid]),2);
	d_inc_sus_k[tid] += thrust::abs(d_store_weighted_sum[tid])*thrust::abs(d_store_weighted_sum[tid]);
}
/*
hipComplex cuExp (hipComplex z){
	float exp_real = expf(z.x)*cosf(z.y);
	float exp_imag = expf(z.x)*sinf(z.y);

	return make_hipComplex(exp_real,exp_imag);
}
*/


int main(int argc, char **argv) {

	// v_d whole lattice
	// black_d black lattice --> white_d white lattice
	unsigned long long *v_d=NULL;
	unsigned long long *black_d=NULL;
	unsigned long long *white_d=NULL;

	// Interaction terms
	unsigned long long *ham_d=NULL;
	unsigned long long *hamB_d=NULL;
	unsigned long long *hamW_d=NULL;

	// Time related stuff
	hipEvent_t start, stop;
    float et;

	// Number of spins per word
	// Bits per word / Bits per Spin
	// 16 spins per word (Theorie)
	const int SPIN_X_WORD = (8*sizeof(*v_d)) / BIT_X_SPIN;

	// Lattice sizes per GPU
	int X = 0;
	int Y = 0;

	// Write results to txt file
	int dumpOut = 0;

	// number of iterations to run
	int nsteps, nwarmup;

	// Random number seed
	unsigned long long seed = 42;

	// number of GPUs
	int ndev = 1;

	// Temperature in absolute units
	float temp  = -1.0f;

	// Probabilties that links connecting any two spins are anti-ferromagnetic
	// Probability for interactions
	int useGenHamilt = 1;
	float hamiltPerc1 = 0.0f;

	// Should we use sublattices or not
	int useSubLatt = 0;

	// Size of sublattices per GPU
	int XSL = 0;
	int YSL = 0;

	// number of sublattices along X and Y per GPU
	int NSLX = 1;
	int NSLY = 1;

	int och;
    while (1) {
        int option_index = 0;
        static struct option long_options[] = {
            {"X", required_argument, 0, 'x'},
            {"Y", required_argument, 0, 'y'},
			{"XSL", required_argument, 0, 1},
			{"YSL", required_argument, 0, 2},
			{"prob", required_argument, 0, 'p'},
			{"nw", required_argument, 0, 'w'},
            {"nit", required_argument, 0, 'n'},
			{"temp", required_argument, 0, 't'},
            {"ndev", required_argument, 0, 'd'},
			{"out", no_argument, 0, 'o'},
            {0, 0, 0, 0}
        };

        och = getopt_long(argc, argv, "x:y:p:w:n:t:do:", long_options, &option_index);
        if (och == -1)
            break;

        switch (och) {
			case   0:// handles long opts with non-NULL flag field
				break;
            case 'x':
				X = atoi(optarg);
                break;
            case 'y':
                Y = atoi(optarg);
                break;
            case 'p':
                hamiltPerc1 = atof(optarg);
                break;
			case 'w':
                nwarmup = atoi(optarg);
                break;
			case 'n':
                nsteps = atoi(optarg);
                break;
			case 't':
				temp = atoi(optarg);
				break;
			case 'd':
				ndev = atoi(optarg);
				break;
			case 'o':
				dumpOut = 1;
				break;
			case 1:
				useSubLatt = 1;
				XSL = atoi(optarg);
				break;
			case 2:
				useSubLatt = 1;
				YSL = atoi(optarg);
				break;
			case '?':
				exit(EXIT_FAILURE);

			default:
				fprintf(stderr, "unknown option: %c\n", och);
				exit(EXIT_FAILURE);
        }
    }

	// check if X or Y are zero
	if (!X || !Y) {
		// check if X is zero
		if (!X) {
			// if Y is not zero and ! Y % 2*S then set x=y
			// X is minimal size 2*SPIN_X_WORD ...
			if (Y && !(Y % (2*SPIN_X_WORD*2*BLOCK_X*BMULT_X))) {
				X = Y;
			}
			// else set X equal to this
			else {
				X = 2*SPIN_X_WORD*2*BLOCK_X*BMULT_X;
			}
		}
		// if Y is zero
		if (!Y) {
			// if x is divisable by BLOCK_Y*BMULT_Y, set Y=X
			if (!(X%(BLOCK_Y*BMULT_Y))) {
				Y = X;
			}
			// else set Y = BLOCK_Y*BMULT_Y
			else {
				Y = BLOCK_Y*BMULT_Y;
			}
		}
	}

	// Check input dimension of X
	if (!X || (X%2) || ((X/2)%(SPIN_X_WORD*2*BLOCK_X*BMULT_X))) {
		fprintf(stderr, "\nPlease specify an X dim multiple of %d\n\n", 2*SPIN_X_WORD*2*BLOCK_X*BMULT_X);
		exit(EXIT_FAILURE);
	}

	// Check input dimension of Y
	if (!Y || (Y%(BLOCK_Y*BMULT_Y))) {
		fprintf(stderr, "\nPlease specify a Y dim multiple of %d\n\n", BLOCK_Y*BMULT_Y);
		exit(EXIT_FAILURE);
	}

	// Check if we want to use sublattices
	if (useSubLatt) {
		// Same as above but for sublattice sizes
		if (!XSL || !YSL) {
			if (!XSL) {
				if (YSL && !(YSL % (2*SPIN_X_WORD*2*BLOCK_X*BMULT_X))) {
					XSL = YSL;
				} else {
					XSL = 2*SPIN_X_WORD*2*BLOCK_X*BMULT_X;
				}
			}
			if (!YSL) {
				if (!(XSL%(BLOCK_Y*BMULT_Y))) {
					YSL = XSL;
				} else {
					YSL = BLOCK_Y*BMULT_Y;
				}
			}
		}

		// X has to be multiple of XSL, XSL has to be even and != 0, XSL multiple of SPIN_X_WORD
		if ((X%XSL) || !XSL || (XSL%2) || ((XSL/2)%(SPIN_X_WORD*2*BLOCK_X*BMULT_X))) {
			fprintf(stderr,
				"\nPlease specify an X sub-lattice dim multiple of %d and divisor of %d\n\n",
				2*SPIN_X_WORD*2*BLOCK_X*BMULT_X, X);
			exit(EXIT_FAILURE);
		}
		// Y multiple of YSL, YSL != 0, Y multiple of Block_Y*..
		if ((Y%YSL) || !YSL || (YSL%(BLOCK_Y*BMULT_Y))) {
			fprintf(stderr,
				"\nPlease specify a Y sub-lattice dim multiple of %d divisor of %d\n\n",
				BLOCK_Y*BMULT_Y, Y);
			exit(EXIT_FAILURE);
		}

		// Set number of Sublattices per GPU
		NSLX = X / XSL;
		NSLY = Y / YSL;
	}

	// If no sublattice
	else {
		// XSL column size, YSL row size of all lattices over all GPUs
		XSL = X;
		YSL = Y*ndev;

		NSLX = 1;
		NSLY = 1;
	}

	// get GPU properties for each GPU
	hipDeviceProp_t props;

	printf("\nUsing GPUs:\n");
	for(int i = 0; i < ndev; i++) {
		CHECK_CUDA(hipGetDeviceProperties(&props, i));
		printf("\t%2d (%s, %d SMs, %d th/SM max, CC %d.%d, ECC %s)\n",
			i, props.name, props.multiProcessorCount,
			props.maxThreadsPerMultiProcessor,
			props.major, props.minor,
			props.ECCEnabled?"on":"off");
	}
	printf("\n");
	// we assums all gpus to be the same so we'll later
	// use the props filled for the last GPU...

	// If we use more than one gpu
	if (ndev > 1) {
		// Check if GPU allows concurrent managed memory access
		for(int i = 0; i < ndev; i++) {
			int attVal = 0;
			CHECK_CUDA(hipDeviceGetAttribute(&attVal, hipDeviceAttributeConcurrentManagedAccess, i));
			if (!attVal) {
				fprintf(stderr,
					"error: device %d does not support concurrent managed memory access!\n", i);
				exit(EXIT_FAILURE);
			}
		}

		// print number of GPUs
		printf("GPUs direct access matrix:\n       ");
		for(int i = 0; i < ndev; i++) {
			printf("%4d", i);
		}

		int missingLinks = 0;
		printf("\n");
		for(int i = 0; i < ndev; i++) {
			printf("GPU %2d:", i);
			// Set index of GPUs
			CHECK_CUDA(hipSetDevice(i));
			for(int j = 0; j < ndev; j++) {
				int access = 1;
				// Check if GPU i can access memory of GPU j
				if (i != j) {
					CHECK_CUDA(hipDeviceCanAccessPeer(&access, i, j));
					// if access, then enable memory access for peer GPU
					if (access) {
						CHECK_CUDA(hipDeviceEnablePeerAccess(j, 0));
					} else {
						missingLinks++;
					}
				}
				printf("%4c", access ? 'V' : 'X');
			}
			printf("\n");
		}
		printf("\n");
		// If missing Links, abort
		if (missingLinks) {
			fprintf(stderr,
				"error: %d direct memory links among devices missing\n",
				missingLinks);
			exit(EXIT_FAILURE);
		}
	}

	// Size of X-dimension of the word lattice per GPU
	size_t lld = (X/2)/SPIN_X_WORD;

	// length of a single color section per GPU (Word lattice)
	size_t llenLoc = static_cast<size_t>(Y)*lld;

	// total lattice length (all GPUs, all colors)
	size_t llen = 2ull*ndev*llenLoc;

	// Create blocks and Threads grid
	// lld/2 Tupel in Initialization?
	dim3 grid(DIV_UP(lld/2, BLOCK_X*BMULT_X),
		  DIV_UP(    Y, BLOCK_Y*BMULT_Y));

	// Creates a CUDA block with Block_X threads in the x dimension and block_Y threads in the y dimension
	dim3 block(BLOCK_X, BLOCK_Y);

	int blocks_per_slx = (XSL/2)/SPIN_X_WORD/2/(BLOCK_X*BMULT_X);
	int blocks_per_sly = YSL/(BLOCK_Y*BMULT_Y);

	// print stuff
	printf("Run configuration:\n");
	printf("\tspin/word: %d\n", SPIN_X_WORD);
	printf("\tspins: %zu\n", llen*SPIN_X_WORD);
	printf("\tseed: %llu\n", seed);
	printf("\titerations: %d\n", nsteps);
	printf("\tblock (X, Y): %d, %d\n", block.x, block.y);
	printf("\ttile  (X, Y): %d, %d\n", BLOCK_X*BMULT_X, BLOCK_Y*BMULT_Y);
	printf("\tgrid  (X, Y): %d, %d\n", grid.x, grid.y);

	printf("\tusing Hamiltonian buffer, setting links to -1 with prob %G\n", hamiltPerc1);

	printf("\n");
	if (useSubLatt) {
		printf("\tusing sub-lattices:\n");
		printf("\t\tno. of sub-lattices per GPU: %8d\n", NSLX*NSLY);
		printf("\t\tno. of sub-lattices (total): %8d\n", ndev*NSLX*NSLY);
		printf("\t\tsub-lattices size:           %7d x %7d\n\n", XSL, YSL);
	}
	printf("\tlocal lattice size:      %8d x %8d\n",      Y, X);
	printf("\ttotal lattice size:      %8d x %8d\n", ndev*Y, X);
	printf("\tlocal lattice shape: 2 x %8d x %8zu (%12zu %s)\n",      Y, lld, llenLoc*2, sizeof(*v_d) == 4 ? "uints" : "ulls");
	printf("\ttotal lattice shape: 2 x %8d x %8zu (%12zu %s)\n", ndev*Y, lld,      llen, sizeof(*v_d) == 4 ? "uints" : "ulls");
	printf("\tmemory: %.2lf MB (%.2lf MB per GPU)\n", (llen*sizeof(*v_d))/(1024.0*1024.0), llenLoc*2*sizeof(*v_d)/(1024.0*1024.0));

	// Maximum block number
	const int redBlocks = MIN(DIV_UP(llen, THREADS),
				  (props.maxThreadsPerMultiProcessor/THREADS)*props.multiProcessorCount);

	// How many spins are up/down
	unsigned long long cntPos;
	unsigned long long cntNeg;

	// pointer array of length MAX_GPU
	unsigned long long *sum_d[MAX_GPU];

	// if only one GPU
	if (ndev == 1) {
		//Allocate memory of size equal to whole lattice and set to 0
		CHECK_CUDA(hipMalloc(&v_d, llen*sizeof(*v_d)));
		CHECK_CUDA(hipMemset(v_d, 0, llen*sizeof(*v_d)));

		// allocate two unsigned long longs
		CHECK_CUDA(hipMalloc(&sum_d[0], 2*sizeof(**sum_d)));

		// Allocate hamiltonian array and set to 0
		CHECK_CUDA(hipMalloc(&ham_d, llen*sizeof(*ham_d)));
		CHECK_CUDA(hipMemset(ham_d, 0, llen*sizeof(*ham_d)));

	// More than one GPU
	} else {
		// Allocate memory accessible by all GPUs
		CHECK_CUDA(hipMallocManaged(&v_d, llen*sizeof(*v_d), hipMemAttachGlobal));
		CHECK_CUDA(hipMallocManaged(&ham_d, llen*sizeof(*ham_d), hipMemAttachGlobal));

		printf("\nSetting up multi-gpu configuration:\n"); fflush(stdout);
		//#pragma omp parallel for schedule(static)

		// Loop over devices
		for(int i = 0; i < ndev; i++) {

			CHECK_CUDA(hipSetDevice(i));

			// Allocate 2 elements for each entry in sum_d and set it to zero
			CHECK_CUDA(hipMalloc(sum_d+i,     2*sizeof(**sum_d)));
        	CHECK_CUDA(hipMemset(sum_d[i], 0, 2*sizeof(**sum_d)));

			// divide v_d into regions for black and white lattices
			CHECK_CUDA(hipMemAdvise(v_d +            i*llenLoc, llenLoc*sizeof(*v_d), hipMemAdviseSetPreferredLocation, i));
			CHECK_CUDA(hipMemAdvise(v_d + (llen/2) + i*llenLoc, llenLoc*sizeof(*v_d), hipMemAdviseSetPreferredLocation, i));

			//Same as above
			CHECK_CUDA(hipMemAdvise(ham_d +            i*llenLoc, llenLoc*sizeof(*ham_d), hipMemAdviseSetPreferredLocation, i));
			CHECK_CUDA(hipMemAdvise(ham_d + (llen/2) + i*llenLoc, llenLoc*sizeof(*ham_d), hipMemAdviseSetPreferredLocation, i));

			// black boundaries up/down
			CHECK_CUDA(hipMemAdvise(v_d +            i*llenLoc,             lld*sizeof(*v_d), hipMemAdviseSetAccessedBy, (i+ndev-1)%ndev));
			CHECK_CUDA(hipMemAdvise(v_d +            i*llenLoc + (Y-1)*lld, lld*sizeof(*v_d), hipMemAdviseSetAccessedBy, (i+ndev+1)%ndev));

			// white boundaries up/down
			CHECK_CUDA(hipMemAdvise(v_d + (llen/2) + i*llenLoc,             lld*sizeof(*v_d), hipMemAdviseSetAccessedBy, (i+ndev-1)%ndev));
			CHECK_CUDA(hipMemAdvise(v_d + (llen/2) + i*llenLoc + (Y-1)*lld, lld*sizeof(*v_d), hipMemAdviseSetAccessedBy, (i+ndev+1)%ndev));

			// Set black/white to all 0s
			CHECK_CUDA(hipMemset(v_d +            i*llenLoc, 0, llenLoc*sizeof(*v_d)));
			CHECK_CUDA(hipMemset(v_d + (llen/2) + i*llenLoc, 0, llenLoc*sizeof(*v_d)));

			CHECK_CUDA(hipMemset(ham_d +            i*llenLoc, 0, llenLoc*sizeof(*ham_d)));
			CHECK_CUDA(hipMemset(ham_d + (llen/2) + i*llenLoc, 0, llenLoc*sizeof(*ham_d)));

			printf("\tGPU %2d done\n", i); fflush(stdout);
		}
	}

	// Set pointer to start of black and white lattice
	black_d = v_d;
	white_d = v_d + llen/2;

	hamB_d = ham_d;
	hamW_d = ham_d + llen/2;

	// Declare two arrays
	float *exp_d[MAX_GPU];
	float  exp_h[2][5];

	// precompute possible exponentials
	// Iterate over all possible spin configurations
	// First loop over spin of interest, either 0 or 1
	// Second loop over all possible up/down configurations of the neighbors
	for(int i = 0; i < 2; i++) {
		for(int j = 0; j < 5; j++) {
			if(temp > 0) {
				exp_h[i][j] = expf((i?-2.0f:2.0f)*static_cast<float>(j*2-4)*(1.0f/temp));
			} else {
				if(j == 2) {
					exp_h[i][j] = 0.5f;
				} else {
					exp_h[i][j] = (i?-2.0f:2.0f)*static_cast<float>(j*2-4);
				}
			}
		}
	}

	// Copy exponentials to GPU
	for(int i = 0; i < ndev; i++) {
		CHECK_CUDA(hipSetDevice(i));
		CHECK_CUDA(hipMalloc(exp_d+i, 2*5*sizeof(**exp_d)));
		CHECK_CUDA(hipMemcpy(exp_d[i], exp_h, 2*5*sizeof(**exp_d), hipMemcpyHostToDevice));
	}

	// Calculate all exp used for weighted summation
	thrust::complex<float> *weighted_exp_d;
	thrust::complex<float> weighted_exp_h[YSL];
	thrust::complex<float> imag = thrust::complex<float>(0,1);

	for(int i = 0; i < YSL; i++) {
		weighted_exp_h[i] = exp(imag*2*M_PI/YSL*i);
	}

	// Copy exponentials to GPU
	for(int i = 0; i < ndev; i++) {
		CHECK_CUDA(hipSetDevice(i));
		CHECK_CUDA(hipMalloc(&weighted_exp_d, YSL*sizeof(*weighted_exp_d)));
		CHECK_CUDA(hipMemcpy(weighted_exp_d, weighted_exp_h, YSL*sizeof(*weighted_exp_d), hipMemcpyHostToDevice));
	}

	// Calculate sums
	int *d_sum_per_block;
	CHECK_CUDA(hipMalloc(&d_sum_per_block, grid.x*grid.y*sizeof(*d_sum_per_block)));

	thrust::complex<float> *d_weighted_sum_per_blocks;
	CHECK_CUDA(hipMalloc(&d_weighted_sum_per_blocks, grid.x*grid.y*sizeof(*d_weighted_sum_per_blocks)));

	int *d_sus_0;
	CHECK_CUDA(hipMalloc(&d_sus_0, NSLX*NSLY*sizeof(*d_sus_0)));
	CHECK_CUDA(hipMemset(d_sus_0, 0, NSLX*NSLY*sizeof(*d_sus_0)));

	float *d_sus_k;
	CHECK_CUDA(hipMalloc(&d_sus_k, NSLX*NSLY*sizeof(*d_sus_k)));
    CHECK_CUDA(hipMemset(d_sus_k, 0, NSLY*NSLX*sizeof(*d_sus_k)));

	/*
	int *d_store_sum;
	CHECK_CUDA(hipMalloc(&d_store_sum, NSLX*NSLY*sizeof(*d_store_sum)));
	CHECK_CUDA(hipMemset(d_store_sum, 0, NSLX*NSLY*sizeof(*d_store_sum)));

	thrust::complex<float> *d_store_weighted_sum;
	CHECK_CUDA(hipMalloc(&d_store_weighted_sum, NSLX*NSLY*sizeof(*d_store_weighted_sum)));
	CHECK_CUDA(hipMemset(d_store_weighted_sum, 0, NSLX*NSLY*sizeof(*d_store_weighted_sum)));

	int *d_inc_sus_0;
	CHECK_CUDA(hipMalloc(&d_inc_sus_0, NSLX*NSLY*sizeof(*d_inc_sus_0)));
	CHECK_CUDA(hipMemset(d_inc_sus_0, 0, NSLX*NSLY*sizeof(*d_inc_sus_0)));

	float *d_inc_sus_k;
	CHECK_CUDA(hipMalloc(&d_inc_sus_k, NSLX*NSLY*sizeof(*d_inc_sus_k)));
	CHECK_CUDA(hipMemset(d_inc_sus_k, 0, NSLX*NSLY*sizeof(*d_inc_sus_k)));
	*/

	// Start and Stop event
	CHECK_CUDA(hipEventCreate(&start));
	CHECK_CUDA(hipEventCreate(&stop));

	for(int i = 0; i < ndev; i++) {

		CHECK_CUDA(hipSetDevice(i));

		// Initialize interaction terms
		hamiltInitB_k<BLOCK_X, BLOCK_Y,
			BMULT_X, BMULT_Y,
			BIT_X_SPIN,
			unsigned long long><<<grid, block>>>(i,
							hamiltPerc1,
							seed+1, // just use a different seed
							i*Y, lld/2,
							reinterpret_cast<ulonglong2 *>(hamB_d));

		hamiltInitW_k<BLOCK_X, BLOCK_Y,
					BMULT_X, BMULT_Y,
					BIT_X_SPIN,
					unsigned long long><<<grid, block>>>((XSL/2)/SPIN_X_WORD/2, YSL, i*Y, lld/2,
									reinterpret_cast<ulonglong2 *>(hamB_d),
									reinterpret_cast<ulonglong2 *>(hamW_d));

		// Init black lattice, lld/2 because of tuples
		latticeInit_k<BLOCK_X, BLOCK_Y,
			      BMULT_X, BMULT_Y,
			      BIT_X_SPIN, C_BLACK,
			      unsigned long long><<<grid, block>>>(i,
								   seed,
								   0, i*Y, lld/2,
								   reinterpret_cast<ulonglong2 *>(black_d));
		CHECK_ERROR("initLattice_k");

		// Init white lattice
		latticeInit_k<BLOCK_X, BLOCK_Y,
			      BMULT_X, BMULT_Y,
			      BIT_X_SPIN, C_WHITE,
			      unsigned long long><<<grid, block>>>(i,
								   seed,
								   0, i*Y, lld/2,
								   reinterpret_cast<ulonglong2 *>(white_d));
		CHECK_ERROR("initLattice_k");
	}


	calculate_average_magnetization<BLOCK_X, BLOCK_Y,
			BMULT_X, BMULT_Y,
			BIT_X_SPIN, unsigned long long><<<grid, block>>>(0,
						XSL, YSL, 0*Y, lld/2,
						reinterpret_cast<ulonglong2 *>(white_d),
						reinterpret_cast<ulonglong2 *>(black_d),
						weighted_exp_d,
						blocks_per_slx,
						blocks_per_sly,
						d_sum_per_block,
						d_weighted_sum_per_blocks);

	clock_t start_timing = clock();

	calculate_incremental_susceptibility<<<1, NSLX*NSLY>>>(blocks_per_slx, blocks_per_sly, NSLX*NSLY, d_sum_per_block, d_weighted_sum_per_blocks, d_sus_0, d_sus_k);

	/*
	for (int i=0; i < NSLX*NSLY; i++){

		if (temp_storage == 0){
			CHECK_CUDA(hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_sum_per_block + i*blocks_per_slx*blocks_per_sly, &d_store_sum[i], blocks_per_slx*blocks_per_sly));
			CHECK_CUDA(hipMalloc(&d_temp, temp_storage));
		}

		if (temp_storage_complex == 0){
			CHECK_CUDA(hipcub::DeviceReduce::Sum(d_temp_complex, temp_storage_complex, d_weighted_sum_per_blocks + i*blocks_per_slx*blocks_per_sly, &d_store_weighted_sum[i], blocks_per_slx*blocks_per_sly));
			CHECK_CUDA(hipMalloc(&d_temp_complex, temp_storage_complex));
		}

		CHECK_CUDA(hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_sum_per_block + i*blocks_per_slx*blocks_per_sly, &d_store_sum[i], blocks_per_slx*blocks_per_sly));
		CHECK_CUDA(hipcub::DeviceReduce::Sum(d_temp_complex, temp_storage_complex, d_weighted_sum_per_blocks + i*blocks_per_slx*blocks_per_sly, &d_store_weighted_sum[i], blocks_per_slx*blocks_per_sly));
	}

	incremental_susceptibility<<<1,NSLX*NSLY>>>(d_store_sum, d_store_weighted_sum, d_inc_sus_0, d_inc_sus_k, NSLX*NSLX);
	*/

	clock_t end_timing = clock();

	double duration = double(end_timing - start_timing) / CLOCKS_PER_SEC;
    std::cout << "Time taken: " << duration << " seconds" << std::endl;

	// Get sum for each sublattice
	int *h_sums_per_block = (int *)malloc(NSLX*NSLY*sizeof(int));
	float *h_weighted_sums_per_block = (float *)malloc(NSLX*NSLY*sizeof(float));

	//CHECK_CUDA(hipMemcpy(h_sums_per_block, d_sus_0, NSLX*NSLY*sizeof(*d_sus_0), hipMemcpyDeviceToHost));
	//CHECK_CUDA(hipMemcpy(h_weighted_sums_per_block, d_sus_k, NSLX*NSLY*sizeof(*d_sus_k), hipMemcpyDeviceToHost));

 	CHECK_CUDA(hipMemcpy(h_sums_per_block, d_sus_0, NSLX*NSLY*sizeof(*d_sus_0), hipMemcpyDeviceToHost));
	CHECK_CUDA(hipMemcpy(h_weighted_sums_per_block, d_sus_k, NSLX*NSLY*sizeof(*d_sus_k), hipMemcpyDeviceToHost));

	for (int i = 0; i < NSLX*NSLY; i++){
		//cout << h_sums_per_block[i] << endl;
		printf("%f\n", h_weighted_sums_per_block[i]);
	}

	// Calculate sum of spins
	//countSpins(ndev, redBlocks, llen, llenLoc, black_d, white_d, sum_d, &cntPos, &cntNeg);

	/*
	printf("\nInitial magnetization: %9.6lf, up_s: %12llu, dw_s: %12llu\n",
	       abs(static_cast<double>(cntPos)-static_cast<double>(cntNeg)) / (llen*SPIN_X_WORD),
	       cntPos, cntNeg);
	*/
	/*
	// Device Synchronize
	for(int i = 0; i < ndev; i++) {
		CHECK_CUDA(hipSetDevice(i));
		CHECK_CUDA(hipDeviceSynchronize());
	}

	// Timing
	double __t0;
	if (ndev == 1) {
		CHECK_CUDA(hipEventRecord(start, 0));
	} else {
		__t0 = Wtime();
	}

	printf("\nPerfom %d Monte Carlo warm up steps \n", nwarmup);
	// Perform Monte Carlo warm-up
	for(int j = 0; j < nwarmup; j++) {
		for(int i = 0; i < ndev; i++) {
			CHECK_CUDA(hipSetDevice(i));
			// Update black lattice
			spinUpdateV_2D_k<BLOCK_X, BLOCK_Y,
					 BMULT_X, BMULT_Y,
					 BIT_X_SPIN, C_BLACK,
					 unsigned long long><<<grid, block>>>(i,
							 		      seed,
									      j+1,
									      (XSL/2)/SPIN_X_WORD/2, YSL,
									      i*Y,  lld/2,
							 		      reinterpret_cast<float (*)[5]>(exp_d[i]),
									      reinterpret_cast<ulonglong2 *>(hamW_d),
									      reinterpret_cast<ulonglong2 *>(white_d),
									      reinterpret_cast<ulonglong2 *>(black_d));
		}

		// Device Synchronize
		if (ndev > 1) {
			for(int i = 0; i < ndev; i++) {
				CHECK_CUDA(hipSetDevice(i));
				CHECK_CUDA(hipDeviceSynchronize());
			}
		}

		// Update white lattice
		for(int i = 0; i < ndev; i++) {
			CHECK_CUDA(hipSetDevice(i));
			spinUpdateV_2D_k<BLOCK_X, BLOCK_Y,
					 BMULT_X, BMULT_Y,
					 BIT_X_SPIN, C_WHITE,
					 unsigned long long><<<grid, block>>>(i,
							 		      seed,
									      j+1,
									      (XSL/2)/SPIN_X_WORD/2, YSL,
									      i*Y, lld/2,
							 		      reinterpret_cast<float (*)[5]>(exp_d[i]),
									      reinterpret_cast<ulonglong2 *>(hamB_d),
									      reinterpret_cast<ulonglong2 *>(black_d),
									      reinterpret_cast<ulonglong2 *>(white_d));
		}

		// Cuda device Synchronize
		if (ndev > 1) {
			for(int i = 0; i < ndev; i++) {
				CHECK_CUDA(hipSetDevice(i));
				CHECK_CUDA(hipDeviceSynchronize());
			}
		}
	}

	printf("\nPerfom %d Monte Carlo steps \n", nsteps);

	// Perform Monte Carlo updates
	for(int j = 0; j < nsteps; j++) {

		for(int i = 0; i < ndev; i++) {
			CHECK_CUDA(hipSetDevice(i));
			// Update black lattice
			spinUpdateV_2D_k<BLOCK_X, BLOCK_Y,
					 BMULT_X, BMULT_Y,
					 BIT_X_SPIN, C_BLACK,
					 unsigned long long><<<grid, block>>>(i,
							 		      seed,
									      j+1,
									      (XSL/2)/SPIN_X_WORD/2, YSL,
									      i*Y, lld/2,
							 		      reinterpret_cast<float (*)[5]>(exp_d[i]),
									      reinterpret_cast<ulonglong2 *>(hamW_d),
									      reinterpret_cast<ulonglong2 *>(white_d),
									      reinterpret_cast<ulonglong2 *>(black_d));
		}

		// Device Synchronize
		if (ndev > 1) {
			for(int i = 0; i < ndev; i++) {
				CHECK_CUDA(hipSetDevice(i));
				CHECK_CUDA(hipDeviceSynchronize());
			}
		}

		// Update white lattice
		for(int i = 0; i < ndev; i++) {
			CHECK_CUDA(hipSetDevice(i));
			spinUpdateV_2D_k<BLOCK_X, BLOCK_Y,
					 BMULT_X, BMULT_Y,
					 BIT_X_SPIN, C_WHITE,
					 unsigned long long><<<grid, block>>>(i,
							 		      seed,
									      j+1,
									      (XSL/2)/SPIN_X_WORD/2, YSL,
									      i*Y, lld/2,
							 		      reinterpret_cast<float (*)[5]>(exp_d[i]),
									      reinterpret_cast<ulonglong2 *>(hamB_d),
									      reinterpret_cast<ulonglong2 *>(black_d),
									      reinterpret_cast<ulonglong2 *>(white_d));
		}

		// Cuda device Synchronize
		if (ndev > 1) {
			for(int i = 0; i < ndev; i++) {
				CHECK_CUDA(hipSetDevice(i));
				CHECK_CUDA(hipDeviceSynchronize());
			}
		}
	}

	// Finish update steps
	if (ndev == 1) {
		CHECK_CUDA(hipEventRecord(stop, 0));
		CHECK_CUDA(hipEventSynchronize(stop));
	}
	else {
		for(int i = 0; i < ndev; i++) {
			CHECK_CUDA(hipSetDevice(i));
			CHECK_CUDA(hipDeviceSynchronize());
		}
		__t0 = Wtime()-__t0;
	}

	// Calculate final magnetization
	countSpins(ndev, redBlocks, llen, llenLoc, black_d, white_d, sum_d, &cntPos, &cntNeg);
	printf("\nFinal   magnetization: %9.6lf, up_s: %12llu, dw_s: %12llu (iter: %8d)\n\n",
	       abs(static_cast<double>(cntPos)-static_cast<double>(cntNeg)) / (llen*SPIN_X_WORD),
	       cntPos, cntNeg, nwarmup + nsteps);


	if (ndev == 1) {
		CHECK_CUDA(hipEventElapsedTime(&et, start, stop));
	} else {
		et = __t0*1.0E+3;
	}


	printf("Kernel execution time for %d update steps: %E ms, %.2lf flips/ns (BW: %.2lf GB/s)\n",
		nsteps+nwarmup, et, static_cast<double>(llen*SPIN_X_WORD)*(nsteps+nwarmup) / (et*1.0E+6),
		//(llen*sizeof(*v_d)*2*j/1.0E+9) / (et/1.0E+3));
		(2ull*(nsteps+nwarmup)*
		 	( sizeof(*v_d)*((llen/2) + (llen/2) + (llen/2)) + // src color read, dst color read, dst color write
			  sizeof(*exp_d)*5*grid.x*grid.y ) /
		1.0E+9) / (et/1.0E+3));
	*/

	// Write lattice
	if (dumpOut) {
		char fname[256];
		snprintf(fname, sizeof(fname), "lattice_%dx%d_T_%f_IT_%08d_", Y, X, temp, nsteps + nwarmup);
		dumpLattice(fname, ndev, Y, lld, llen, llenLoc, v_d);
	}

	// free memory for all GPUs and stuff
	CHECK_CUDA(hipFree(v_d));
	if (useGenHamilt) {
		CHECK_CUDA(hipFree(ham_d));
	}
	if (ndev == 1) {
		CHECK_CUDA(hipFree(exp_d[0]));
		CHECK_CUDA(hipFree(sum_d[0]));
	} else {
		for(int i = 0; i < ndev; i++) {
			CHECK_CUDA(hipSetDevice(i));
			CHECK_CUDA(hipFree(exp_d[i]));
			CHECK_CUDA(hipFree(sum_d[i]));
		}
	}
	for(int i = 0; i < ndev; i++) {
        CHECK_CUDA(hipSetDevice(i));
        CHECK_CUDA(hipDeviceReset());
    }

	return 0;
}
