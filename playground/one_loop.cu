#include "hip/hip_runtime.h"
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <time.h>
#include <hiprand.h>
#include <hipcub/hipcub.hpp>
#include <thrust/complex.h>
#include <cmath>

using namespace std;

#define THREADS 128

// Write interaction bonds to file
void write_bonds(signed char* interactions, std::string filename, long long nx, long long ny){
    printf("Writing bonds to %s ...\n", filename.c_str());
    signed char *interactions_host;
    interactions_host = (signed char*)malloc(2*nx*ny*sizeof(*interactions_host));
    hipMemcpy(interactions_host,interactions, 2*nx*ny*sizeof(*interactions), hipMemcpyDeviceToHost);
        
      std::ofstream f;
      f.open(filename);
      if (f.is_open()) {
        for (int i = 0; i < 2*nx; i++) {
          for (int j = 0; j < ny; j++) {
             f << (int)interactions_host[i * ny + j] << " ";
          }
          f << std::endl;
        }
      }
      f.close();
      hipFree(interactions);
      free(interactions_host);
}

__global__ void init_randombond(signed char* interactions, const float* __restrict__ interaction_randvals,
    const long long nx, const long long ny, const float p){
        
        const long long tid = static_cast<long long>(threadIdx.x + blockIdx.x * blockDim.x);
        
        if (tid >= 2*nx*ny) return;

        float bondrandval = interaction_randvals[tid];
        signed char bondval = (bondrandval<p)? -1 : 1;
        interactions[tid] = bondval;                                  
}

void init_interactions_with_seed(signed char* interactions, const long long seed, const long long nx, const long long ny, const float p){
    int blocks = (nx*ny*2 + THREADS -1)/THREADS;

    //Setup cuRAND generator for the random bond sign
    hiprandGenerator_t interaction_rng;
    hiprandCreateGenerator(&interaction_rng,HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
    hiprandSetPseudoRandomGeneratorSeed(interaction_rng,seed);
    
    float *interaction_randvals;
    hipMalloc(&interaction_randvals,nx*ny*2*sizeof(*interaction_randvals));

    hiprandGenerateUniform(interaction_rng,interaction_randvals,nx*ny*2);
    init_randombond<<<blocks, THREADS>>>(interactions, interaction_randvals,nx,ny,p);
    
    hipFree(interaction_randvals); 
}

// Initialize lattice spins
__global__ void init_spins(signed char* lattice, const float* __restrict__ randvals,
    const long long nx, const long long ny) {
        const long long  tid = static_cast<long long>(blockDim.x) * blockIdx.x + threadIdx.x;
        if (tid >= nx * ny) return;
        
        float randval = randvals[tid];
        signed char val = (randval < 0.5f) ? -1 : 1;
        lattice[tid] = val;
}

void init_spins_with_seed(signed char* lattice_b, signed char* lattice_w, const long long seed, const long long nx, const long long ny){
    
    int blocks = (nx*ny*2 + THREADS -1)/THREADS;
    
    // Setup cuRAND generator
    hiprandGenerator_t rng;
    hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
    hiprandSetPseudoRandomGeneratorSeed(rng, seed);

    float *randvals;
    hipMalloc(&randvals, nx * ny/2 * sizeof(*randvals));

    //Initialize the arrays for white and black lattice
    hiprandGenerateUniform(rng, randvals, nx*ny/2);
    init_spins<<<blocks, THREADS>>>(lattice_b, randvals, nx, ny/2);
    hiprandGenerateUniform(rng, randvals, nx*ny/2);
    init_spins<<<blocks, THREADS>>>(lattice_w, randvals, nx, ny/2);

    hipFree(randvals); 
}

__global__ void B2_lattices(signed char* lattice_b, signed char* lattice_w, const float *wave_vector, thrust::complex<float> *sum,  int nx, int ny){
    
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    int i = tid/ny;
    int j = tid%ny;

    if (i>=nx || j >= ny) return;

    int b_orig_j;
    int w_orig_j; 

    if (i%2==0){
        b_orig_j = 2*j +1;
        w_orig_j = 2*j;
    }
    else{
        b_orig_j = 2*j;
        w_orig_j = 2*j + 1;
    }

    thrust::complex<float> imag = thrust::complex<float>(0, 1.0f);

    float dot_b = wave_vector[0]*i + wave_vector[1]*b_orig_j;
    float dot_w = wave_vector[0]*i + wave_vector[1]*w_orig_j;
    sum[tid] = lattice_b[tid]*exp(imag*dot_b) + lattice_w[tid]*exp(imag*dot_w);
}

template<bool is_black>
__global__ void calc_energy(float* sum, signed char* lattice, signed char* __restrict__ op_lattice, 
    signed char* interactions, const long long nx, const long long ny, const float coupling_constant){
    
    const long long tid = static_cast<long long>(blockDim.x)*blockIdx.x + threadIdx.x;
    const int i = tid/ny;
    const int j = tid%ny;
  
    if (i>=nx || j >= ny) return;

        // Set up periodic boundary conditions
    int ipp = (i + 1 < nx) ? i + 1 : 0;
    int inn = (i - 1 >= 0) ? i - 1: nx - 1;
    int jpp = (j + 1 < ny) ? j + 1 : 0;
    int jnn = (j - 1 >= 0) ? j - 1: ny - 1;

    int joff;
    int jcouplingoff;
    int icouplingpp;
    int icouplingnn;

    if (is_black) {
        icouplingpp = 2*(nx-1)*ny + 2*(ny*(i+1) + j) + (i+1)%2;
        icouplingnn = 2*(nx-1)*ny + 2*(ny*(inn+1) + j) + (i+1)%2;
        joff = (i % 2) ? jnn : jpp;

        if (i % 2) {
            jcouplingoff = 2 * (i * ny + joff) + 1;
        } else {
            if (j + 1 >= ny) {
                jcouplingoff = 2 * (i * ny + j + 1) - 1;
            } else {
                jcouplingoff = 2 * (i * ny + joff) - 1;
            }
        }

    } else {
        icouplingpp = 2*(nx-1)*ny + 2*(ny*(i+1) + j) + i%2;
        icouplingnn = 2*(nx-1)*ny + 2*(ny*(inn+1) + j) + i%2;
        joff = (i % 2) ? jpp : jnn;

        if (i % 2) {
            if (j+1 >= ny) {
                jcouplingoff = 2 * (i * ny + j + 1) - 1;
            } else {
                jcouplingoff = 2 * (i * ny + joff) - 1;
            }
        } else {
            jcouplingoff = 2 * (i * ny + joff) + 1;
        }
    }

    // Compute sum of nearest neighbor spins times the coupling
    sum[tid] = -1 * coupling_constant*lattice[i*ny+j]*(op_lattice[inn * ny + j]*interactions[icouplingnn] + op_lattice[i * ny + j]*interactions[2*(i*ny + j)] 
               + op_lattice[ipp * ny + j]*interactions[icouplingpp] + op_lattice[i * ny + joff]*interactions[jcouplingoff]);
}

__global__ void abs_square(thrust::complex<float> *d_store_sum, const int num_iterations){
    const long long tid = static_cast<long long>(blockDim.x)*blockIdx.x + threadIdx.x;

    if (tid >= num_iterations) return;

    d_store_sum[tid] = thrust::abs(d_store_sum[tid]) * thrust::abs(d_store_sum[tid]);
}

__global__ void exp_beta(float *d_store_energy, float inv_temp, const int num_iterations){
    
    const long long tid = static_cast<long long>(blockDim.x)*blockIdx.x + threadIdx.x;

    if (tid >= num_iterations) return;

    d_store_energy[tid] = exp(-inv_temp*d_store_energy[tid]);
}

 // Write lattice configuration to file
 void write_lattice(signed char *lattice_b, signed char *lattice_w, std::string filename, long long nx, long long ny) {
    printf("Writing lattice to %s...\n", filename.c_str());
    signed char *lattice_h, *lattice_b_h, *lattice_w_h;
    lattice_h = (signed char*) malloc(nx * ny * sizeof(*lattice_h));
    lattice_b_h = (signed char*) malloc(nx * ny/2 * sizeof(*lattice_b_h));
    lattice_w_h = (signed char*) malloc(nx * ny/2 * sizeof(*lattice_w_h));
  
    hipMemcpy(lattice_b_h, lattice_b, nx * ny/2 * sizeof(*lattice_b), hipMemcpyDeviceToHost);
    hipMemcpy(lattice_w_h, lattice_w, nx * ny/2 * sizeof(*lattice_w), hipMemcpyDeviceToHost);
  
    for (int i = 0; i < nx; i++) {
      for (int j = 0; j < ny/2; j++) {
        if (i % 2) {
          lattice_h[i*ny + 2*j+1] = lattice_w_h[i*ny/2 + j];
          lattice_h[i*ny + 2*j] = lattice_b_h[i*ny/2 + j];
        } else {
          lattice_h[i*ny + 2*j] = lattice_w_h[i*ny/2 + j];
          lattice_h[i*ny + 2*j+1] = lattice_b_h[i*ny/2 + j];
        }
      }
    }
  
    std::ofstream f;
    f.open(filename);
    if (f.is_open()) {
      for (int i = 0; i < nx; i++) {
        for (int j = 0; j < ny; j++) {
           f << (int)lattice_h[i * ny + j] << " ";
        }
        f << std::endl;
      }
    }
    f.close();
  
    free(lattice_h);
    free(lattice_b_h);
    free(lattice_w_h);
}

void calculate_B2(signed char *lattice_b, signed char *lattice_w, thrust::complex<float> *d_store_sum, float *d_wave_vector, int i, const long nx, const long ny){
    // Calculate B2 and reduce sum
    int blocks = (nx*ny*2 + THREADS -1)/THREADS;

    thrust::complex<float> *d_sum;
    hipMalloc(&d_sum, nx*ny/2*sizeof(*d_sum));

    B2_lattices<<<blocks, THREADS>>>(lattice_b, lattice_w, d_wave_vector, d_sum, nx, ny/2);

    // Run sum-reduction
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, &d_store_sum[i], nx*ny/2);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, &d_store_sum[i], nx*ny/2);

    hipFree(d_sum);
}

void calculate_energy(signed char *lattice_b, signed char *lattice_w, signed char *d_interactions, float *d_store_energy, float coupling_constant, int i, long nx, long ny){
    // Calculate energy and reduce sum
    int blocks = (nx*ny*2 + THREADS -1)/THREADS;
    
    float *d_energy;
    hipMalloc(&d_energy, nx*ny/2*sizeof(*d_energy));

    calc_energy<true><<<blocks,THREADS>>>(d_energy, lattice_b, lattice_w, d_interactions, nx, ny/2, coupling_constant);

    // Run sum-reduction
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_energy, &d_store_energy[i], nx*ny/2);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_energy, &d_store_energy[i], nx*ny/2);
    
    hipFree(d_energy);
}

__global__ void weighted_energies(float *d_weighted_energies, float *d_store_energy, thrust::complex<float> *d_store_sum, float *d_partition_function, const int num_iterations){
    const long long tid = static_cast<long long>(blockDim.x)*blockIdx.x + threadIdx.x;

    if (tid >= num_iterations) return;

    d_weighted_energies[tid] = d_store_energy[tid]*d_store_sum[tid].real() / *d_partition_function;
}

void calculate_weighted_energies(float *d_error_weight, float *d_store_energy, thrust::complex<float> *d_store_sum, float *d_partition_function, const int num_iterations, const int blocks){
    // Calculate energy and reduce sum

    float *d_weighted_energies;
    hipMalloc(&d_weighted_energies, num_iterations*sizeof(*d_weighted_energies));

    weighted_energies<<<blocks, THREADS>>>(d_weighted_energies, d_store_energy, d_store_sum, d_partition_function, num_iterations);

    // Run sum-reduction
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_weighted_energies, d_error_weight, num_iterations);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_weighted_energies, d_error_weight, num_iterations);
    
    hipFree(d_weighted_energies);
}

template<bool is_black>
__global__ void update_lattice(signed char* lattice, signed char* __restrict__ op_lattice, const float* __restrict__ randvals, signed char* interactions,
                               const float inv_temp,
                               const long long nx,
                               const long long ny,
                               const float coupling_constant) {

    const long long tid = static_cast<long long>(blockDim.x)*blockIdx.x + threadIdx.x;
    
    const int i = tid/ny;
    const int j = tid%ny;

    if (i>=nx || j >= ny) return;

    // Set up periodic boundary conditions
    int ipp = (i + 1 < nx) ? i + 1 : 0;
    int inn = (i - 1 >= 0) ? i - 1: nx - 1;
    int jpp = (j + 1 < ny) ? j + 1 : 0;
    int jnn = (j - 1 >= 0) ? j - 1: ny - 1;

    int joff;
    int jcouplingoff;
    int icouplingpp;
    int icouplingnn;

    if (is_black) {
        icouplingpp = 2*(nx-1)*ny + 2*(ny*(i+1) + j) + (i+1)%2;
        icouplingnn = 2*(nx-1)*ny + 2*(ny*(inn+1) + j) + (i+1)%2;
        joff = (i % 2) ? jnn : jpp;

        if (i % 2) {
            jcouplingoff = 2 * (i * ny + joff) + 1;
        } else {
            if (j + 1 >= ny) {
                jcouplingoff = 2 * (i * ny + j + 1) - 1;
            } else {
                jcouplingoff = 2 * (i * ny + joff) - 1;
            }
        }
    } else {
        icouplingpp = 2*(nx-1)*ny + 2*(ny*(i+1) + j) + i%2;
        icouplingnn = 2*(nx-1)*ny + 2*(ny*(inn+1) + j) + i%2;
        joff = (i % 2) ? jpp : jnn;

        if (i % 2) {
            if (j+1 >= ny) {
                jcouplingoff = 2 * (i * ny + j + 1) - 1;
            } else {
                jcouplingoff = 2 * (i * ny + joff) - 1;
            }
        } else {
            jcouplingoff = 2 * (i * ny + joff) + 1;
        }
    }

    // Compute sum of nearest neighbor spins times the coupling
    signed char nn_sum = op_lattice[inn * ny + j]*interactions[icouplingnn] + op_lattice[i * ny + j]*interactions[2*(i*ny + j)] 
                        + op_lattice[ipp * ny + j]*interactions[icouplingpp] + op_lattice[i * ny + joff]*interactions[jcouplingoff];

    // Compute sum of nearest neighbor spins
    //signed char nn_sum = op_lattice[inn * ny + j] + op_lattice[i * ny + j] + op_lattice[ipp * ny + j] + op_lattice[i * ny + joff];

    // Determine whether to flip spin
    signed char lij = lattice[i * ny + j];
    float acceptance_ratio = exp(-2 * coupling_constant * nn_sum * lij);
    if (randvals[i*ny + j] < acceptance_ratio) {
        lattice[i * ny + j] = -lij;
    }  
}

void update(signed char *lattice_b, signed char *lattice_w, float* randvals, hiprandGenerator_t rng, signed char* interactions, float inv_temp, long long nx, long long ny, float coupling_constant) {
 
    // Setup CUDA launch configuration
    int blocks = (nx * ny/2 + THREADS - 1) / THREADS;

    // Update black
    hiprandGenerateUniform(rng, randvals, nx*ny/2);
    update_lattice<true><<<blocks, THREADS>>>(lattice_b, lattice_w, randvals,interactions, inv_temp, nx, ny/2,coupling_constant);

    // Update white
    hiprandGenerateUniform(rng, randvals, nx*ny/2);
    update_lattice<false><<<blocks, THREADS>>>(lattice_w, lattice_b, randvals,interactions, inv_temp, nx, ny/2, coupling_constant);
}

int main(void){
    // Initialize all possible parameters
    int niters = 1000;
    int nwarmup = 100;
    long nx = 1000;
    long ny = 1000;  
    //float p = 0.15;
    float p = 0.031091730001f;
    float alpha = 1.0f;
    float TCRIT = 8.0f;
    float inv_temp = 1.0f / (alpha*TCRIT);
    const float coupling_constant = 0.5*TCRIT*log((1-p)/p);
  
    int num_iterations = 2;
    
    int blocks = (nx*ny*2 + THREADS -1)/THREADS;

    // Initialize seeds used for spin and interaction initialization
    unsigned long long seeds_spins = 0ULL;
    unsigned long long seeds_interactions = 0ULL;
    
    // Allocate the wave vector and copy it to GPU memory
    float wave_vector[2] = {0,0};

    float *d_wave_vector;
    hipMalloc(&d_wave_vector, 2 * sizeof(*d_wave_vector));
    hipMemcpy(d_wave_vector, wave_vector, 2*sizeof(float), hipMemcpyHostToDevice);

    // Initialize arrays on the GPU to store results per spin system for energy and sum of B2
    thrust::complex<float> *d_store_sum;
    hipMalloc(&d_store_sum, num_iterations*sizeof(*d_store_sum));

    float *d_store_energy;
    hipMalloc(&d_store_energy, num_iterations*sizeof(*d_store_energy));

    //Setup interaction lattice on device
    signed char *d_interactions;
    hipMalloc(&d_interactions, nx*ny*2*sizeof(*d_interactions));

    init_interactions_with_seed(d_interactions, seeds_interactions, nx, ny, p);
    
    //Synchronize devices
    hipDeviceSynchronize();

    // Loop over number of iterations
    for (int i=0; i<num_iterations; i++){

        // Setup black and white lattice arrays on device
        signed char *lattice_b, *lattice_w;
        hipMalloc(&lattice_b, nx * ny/2 * sizeof(*lattice_b));
        hipMalloc(&lattice_w, nx * ny/2 * sizeof(*lattice_w));

        init_spins_with_seed(lattice_b, lattice_w, seeds_spins, nx, ny);

        // Setup cuRAND generator
        hiprandGenerator_t rng;
        hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
        hiprandSetPseudoRandomGeneratorSeed(rng, seeds_spins);
        float *randvals;
        hipMalloc(&randvals, nx * ny/2 * sizeof(*randvals));

        //Synchronize devices
        hipDeviceSynchronize();
        
        // Warmup iterations
        printf("Starting warmup...\n");
        for (int j = 0; j < nwarmup; j++) {
            update(lattice_b, lattice_w, randvals, rng, d_interactions, inv_temp, nx, ny, coupling_constant);
        }

        //Synchronize devices
        hipDeviceSynchronize();

        for (int j = 0; j < niters; j++) {
            update(lattice_b, lattice_w, randvals, rng, d_interactions, inv_temp, nx, ny,coupling_constant);
            if (j % 1000 == 0) printf("Completed %d/%d iterations...\n", j+1, niters);
        }

        hipDeviceSynchronize();
        
        calculate_B2(lattice_b, lattice_w, d_store_sum, d_wave_vector, i, nx, ny);

        hipDeviceSynchronize();

        calculate_energy(lattice_b, lattice_w, d_interactions, d_store_energy, coupling_constant, i, nx, ny);

        hipDeviceSynchronize();

        write_lattice(lattice_b, lattice_w, "final_lattice_" + std::to_string(i) + ".txt", nx, ny);
    }

    float *h_energy = (float *)malloc(num_iterations*sizeof(float));

    hipMemcpy(h_energy, d_store_energy, num_iterations*sizeof(float), hipMemcpyDeviceToHost);

    for (int j=0; j<num_iterations;j++){
        cout << h_energy[j];
    }
    
    write_bonds(d_interactions, "final_bonds.txt", nx, ny);
    
    /*
    abs_square<<<blocks, THREADS>>>(d_store_sum, num_iterations); 
    exp_beta<<<blocks, THREADS>>>(d_store_energy, inv_temp, num_iterations);

    // Calculate partition function
    float *d_partition_function;
    hipMalloc(&d_partition_function, sizeof(float));
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_store_energy, d_partition_function, num_iterations);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_store_energy, d_partition_function, num_iterations);
    
    float *d_error_weight;
    hipMalloc(&d_error_weight, sizeof(*d_error_weight));

    calculate_weighted_energies(d_error_weight, d_store_energy, d_store_sum, d_partition_function, num_iterations, blocks);
    
    float *h_error_weight = (float *)malloc(sizeof(float));
    hipMemcpy(h_error_weight, d_error_weight, sizeof(float), hipMemcpyDeviceToHost);

    cout << *h_error_weight;
    */
}