#include <cstdlib>
#include <iostream>
#include <time.h>
#include <fstream>
#include <hiprand.h>
#include <hipcub/hipcub.hpp>
#include <thrust/complex.h>
#include <vector>

using namespace std;

#define THREADS 128

void sum_with_index(float *d_array, int num_values, float* d_res, int i){
  // Variables used for sum reduction
  void *d_temp = NULL;
  size_t temp_storage = 0;

  if (i==-1){
    // Sum reduction
    hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_array, d_res, num_values);
    hipMalloc(&d_temp, temp_storage);
    hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_array, d_res, num_values);
  }
  else{
    // Sum reduction
    hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_array, &d_res[i], num_values);
    hipMalloc(&d_temp, temp_storage);
    hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_array, &d_res[i], num_values);
  }
  
}

float* sum_one(float *d_array, int num_values){
  // Variables used for sum reduction
  void *d_temp = NULL;
  size_t temp_storage = 0;

  float *d_sum;
  hipMalloc(&d_sum, sizeof(float));
  
  // Sum reduction
  hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_array, d_sum, num_values);
  hipMalloc(&d_temp, temp_storage);
  hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_array, d_sum, num_values);
  
  return d_sum;
}

int main(int argc, char **argv) {

  int num_values = 10;
  
  float *h_array = (float *)malloc(num_values*sizeof(float));

  for (int i=0; i<num_values; i++){
    h_array[i] = i;
  }

  float *d_array;
  hipMalloc(&d_array, num_values*sizeof(float));
  
  hipMemcpy(d_array, h_array, num_values*sizeof(float), hipMemcpyHostToDevice);

  float* d_res_single;
  hipMalloc(&d_res, num_values*sizeof(float));
  
  sum_with_index(d_array, num_values, d_res, -1);
  
  float *h_sum = (float *)malloc(num_values*sizeof(float));
  hipMemcpy(h_sum, d_res, num_values*sizeof(float), hipMemcpyDeviceToHost);

  printf("%f", h_sum[0]);
}