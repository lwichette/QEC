#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <time.h>
#include <fstream>
#include <hiprand.h>
#include <hipcub/hipcub.hpp>
#include <thrust/complex.h>
#include <vector>
#include <sys/stat.h>

using namespace std;

#define THREADS 128

void sum_with_index(float *d_array, int num_values, float* d_res, int i){
  // Variables used for sum reduction
  void *d_temp = NULL;
  size_t temp_storage = 0;

  if (i==-1){
    // Sum reduction
    hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_array, d_res, num_values);
    hipMalloc(&d_temp, temp_storage);
    hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_array, d_res, num_values);
  }
  else{
    // Sum reduction
    hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_array, &d_res[i], num_values);
    hipMalloc(&d_temp, temp_storage);
    hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_array, &d_res[i], num_values);
  }
}

float* sum_one(float *d_array, int num_values){
  // Variables used for sum reduction
  void *d_temp = NULL;
  size_t temp_storage = 0;

  float *d_sum;
  hipMalloc(&d_sum, sizeof(float));
  
  // Sum reduction
  hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_array, d_sum, num_values);
  hipMalloc(&d_temp, temp_storage);
  hipcub::DeviceReduce::Sum(d_temp, temp_storage, d_array, d_sum, num_values);
  
  return d_sum;
}

void create_array(int n){
  std::vector<float> test(n);
}

__global__ void initialize(float *d_test, int num_values){
  
  const long long tid = static_cast<long long>(threadIdx.x + blockIdx.x * blockDim.x);

  if (tid>=num_values) return;

  d_test[tid] = 3000;

}
int main(int argc, char **argv) {
  char* results = "results/test3";

  // Structure which would store the metadata
  struct stat sb;

  if (stat(results, &sb) == 0){
      cout << "Results already exist, check file name";
      return 0;
  }
  else{
      mkdir(results, S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
  }
}