#include "hip/hip_runtime.h"
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <time.h>
#include <hiprand.h>
#include <hipcub/hipcub.hpp>
#include <thrust/complex.h>
#include <cmath>

using namespace std;

#define THREADS 128


void write_bonds(signed char* interactions, std::string filename, long long nx, long long ny){
    printf("Writing bonds to %s ...\n", filename.c_str());
    signed char *interactions_host;
    interactions_host = (signed char*)malloc(2*nx*ny*sizeof(*interactions_host));
    hipMemcpy(interactions_host,interactions, 2*nx*ny*sizeof(*interactions), hipMemcpyDeviceToHost);
        
      std::ofstream f;
      f.open(filename);
      if (f.is_open()) {
        for (int i = 0; i < 2*nx; i++) {
          for (int j = 0; j < ny; j++) {
             f << (int)interactions_host[i * ny + j] << " ";
          }
          f << std::endl;
        }
      }
      f.close();
      hipFree(interactions);
      free(interactions_host);
}

void write_lattice(signed char *lattice_b, signed char *lattice_w, std::string filename, long long nx, long long ny) {
    printf("Writing lattice to %s...\n", filename.c_str());
    signed char *lattice_h, *lattice_b_h, *lattice_w_h;
    lattice_h = (signed char*) malloc(nx * ny * sizeof(*lattice_h));
    lattice_b_h = (signed char*) malloc(nx * ny/2 * sizeof(*lattice_b_h));
    lattice_w_h = (signed char*) malloc(nx * ny/2 * sizeof(*lattice_w_h));
  
    hipMemcpy(lattice_b_h, lattice_b, nx * ny/2 * sizeof(*lattice_b), hipMemcpyDeviceToHost);
    hipMemcpy(lattice_w_h, lattice_w, nx * ny/2 * sizeof(*lattice_w), hipMemcpyDeviceToHost);
  
    for (int i = 0; i < nx; i++) {
      for (int j = 0; j < ny/2; j++) {
        if (i % 2) {
          lattice_h[i*ny + 2*j+1] = lattice_b_h[i*ny/2 + j];
          lattice_h[i*ny + 2*j] = lattice_w_h[i*ny/2 + j];
        } else {
          lattice_h[i*ny + 2*j] = lattice_b_h[i*ny/2 + j];
          lattice_h[i*ny + 2*j+1] = lattice_w_h[i*ny/2 + j];
        }
      }
    }
  
    std::ofstream f;
    f.open(filename);
    if (f.is_open()) {
      for (int i = 0; i < nx; i++) {
        for (int j = 0; j < ny; j++) {
           f << (int)lattice_h[i * ny + j] << " ";
        }
        f << std::endl;
      }
    }
    f.close();
  
    free(lattice_h);
    free(lattice_b_h);
    free(lattice_w_h);
}

__global__ void init_randombond(signed char* interactions, const float* __restrict__ interaction_randvals,
    const long long nx, const long long ny, const float p){
        
        const long long tid = static_cast<long long>(threadIdx.x + blockIdx.x * blockDim.x);
        
        if (tid >= 2*nx*ny) return;

        float bondrandval = interaction_randvals[tid];
        signed char bondval = (bondrandval<p)? -1 : 1;
        interactions[tid] = bondval;                                  
}

void init_interactions_with_seed(signed char* interactions, const long long seed, const long long nx, const long long ny, const float p){
    int blocks = (nx*ny*2 + THREADS -1)/THREADS;

    //Setup cuRAND generator for the random bond sign
    hiprandGenerator_t interaction_rng;
    hiprandCreateGenerator(&interaction_rng,HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
    hiprandSetPseudoRandomGeneratorSeed(interaction_rng,seed);
    
    float *interaction_randvals;
    hipMalloc(&interaction_randvals,nx*ny*2*sizeof(*interaction_randvals));

    hiprandGenerateUniform(interaction_rng,interaction_randvals,nx*ny*2);
    init_randombond<<<blocks, THREADS>>>(interactions, interaction_randvals,nx,ny,p);
    
    hipFree(interaction_randvals); 
}

__global__ void init_spins(signed char* lattice, const float* __restrict__ randvals,
    const long long nx, const long long ny) {
        const long long  tid = static_cast<long long>(blockDim.x) * blockIdx.x + threadIdx.x;
        if (tid >= nx * ny) return;
        
        float randval = randvals[tid];
        signed char val = (randval < 0.5f) ? -1 : 1;
        lattice[tid] = val;
}

void init_spins_with_seed(signed char* lattice_b, signed char* lattice_w, const long long seed, const long long nx, const long long ny){
    
    int blocks = (nx*ny*2 + THREADS -1)/THREADS;
    
    // Setup cuRAND generator
    hiprandGenerator_t rng;
    hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
    hiprandSetPseudoRandomGeneratorSeed(rng, seed);

    float *randvals;
    hipMalloc(&randvals, nx * ny/2 * sizeof(*randvals));

    //Initialize the arrays for white and black lattice
    hiprandGenerateUniform(rng, randvals, nx*ny/2);
    init_spins<<<blocks, THREADS>>>(lattice_b, randvals, nx, ny/2);
    hiprandGenerateUniform(rng, randvals, nx*ny/2);
    init_spins<<<blocks, THREADS>>>(lattice_w, randvals, nx, ny/2);

    hipFree(randvals); 
}

template<bool is_black>
__global__ void update_lattice(signed char* lattice, signed char* __restrict__ op_lattice, const float* __restrict__ randvals, signed char* interactions,
                               const float inv_temp,
                               const long long nx,
                               const long long ny,
                               const float coupling_constant) {

    const long long tid = static_cast<long long>(blockDim.x)*blockIdx.x + threadIdx.x;
    
    const int i = tid/ny;
    const int j = tid%ny;

    if (i>=nx || j >= ny) return;

    // Set up periodic boundary conditions
    int ipp = (i + 1 < nx) ? i + 1 : 0;
    int inn = (i - 1 >= 0) ? i - 1: nx - 1;
    int jpp = (j + 1 < ny) ? j + 1 : 0;
    int jnn = (j - 1 >= 0) ? j - 1: ny - 1;

    int joff;
    int jcouplingoff;
    int icouplingpp;
    int icouplingnn;

    if (is_black) {
        icouplingpp = 2*(nx-1)*ny + 2*(ny*(i+1) + j) + (i+1)%2;
        icouplingnn = 2*(nx-1)*ny + 2*(ny*(inn+1) + j) + (i+1)%2;
        joff = (i % 2) ? jnn : jpp;

        if (i % 2) {
            jcouplingoff = 2 * (i * ny + joff) + 1;
        } else {
            if (j + 1 >= ny) {
                jcouplingoff = 2 * (i * ny + j + 1) - 1;
            } else {
                jcouplingoff = 2 * (i * ny + joff) - 1;
            }
        }
    } else {
        icouplingpp = 2*(nx-1)*ny + 2*(ny*(i+1) + j) + i%2;
        icouplingnn = 2*(nx-1)*ny + 2*(ny*(inn+1) + j) + i%2;
        joff = (i % 2) ? jpp : jnn;

        if (i % 2) {
            if (j+1 >= ny) {
                jcouplingoff = 2 * (i * ny + j + 1) - 1;
            } else {
                jcouplingoff = 2 * (i * ny + joff) - 1;
            }
        } else {
            jcouplingoff = 2 * (i * ny + joff) + 1;
        }
    }

    // Compute sum of nearest neighbor spins times the coupling
    signed char nn_sum = op_lattice[inn * ny + j]*interactions[icouplingnn] + op_lattice[i * ny + j]*interactions[2*(i*ny + j)] 
                        + op_lattice[ipp * ny + j]*interactions[icouplingpp] + op_lattice[i * ny + joff]*interactions[jcouplingoff];

    // Compute sum of nearest neighbor spins
    //signed char nn_sum = op_lattice[inn * ny + j] + op_lattice[i * ny + j] + op_lattice[ipp * ny + j] + op_lattice[i * ny + joff];

    // Determine whether to flip spin
    signed char lij = lattice[i * ny + j];
    float acceptance_ratio = exp(-2 * coupling_constant * nn_sum * lij);
    if (randvals[i*ny + j] < acceptance_ratio) {
        lattice[i * ny + j] = -lij;
    }  
}

void update(signed char *lattice_b, signed char *lattice_w, float* randvals, hiprandGenerator_t rng, signed char* interactions, float inv_temp, long long nx, long long ny, float coupling_constant) {
 
    // Setup CUDA launch configuration
    int blocks = (nx * ny/2 + THREADS - 1) / THREADS;

    // Update black
    hiprandGenerateUniform(rng, randvals, nx*ny/2);
    update_lattice<true><<<blocks, THREADS>>>(lattice_b, lattice_w, randvals,interactions, inv_temp, nx, ny/2,coupling_constant);

    // Update white
    hiprandGenerateUniform(rng, randvals, nx*ny/2);
    update_lattice<false><<<blocks, THREADS>>>(lattice_w, lattice_b, randvals,interactions, inv_temp, nx, ny/2, coupling_constant);
}

int main(int argc, char **argv) {
    // Initialize all possible parameters
    int niters = 1000;
    int nwarmup = 100;
    long nx = 1000;
    long ny = 1000;  
    //float p = 0.15;
    float p = 0.031091730001f;
    float alpha = 1.0f;
    float TCRIT = 8.0f;
    float inv_temp = 1.0f / (alpha*TCRIT);
    const float coupling_constant = 0.5*TCRIT*log((1-p)/p);

    int num_iterations = 1;
    
    int blocks = (nx*ny*2 + THREADS -1)/THREADS;

    // Initialize seeds used for spin and interaction initialization
    unsigned long long seeds_spins = 1234ULL;
    unsigned long long seeds_interactions =  1234ULL;
    
    // Allocate the wave vector and copy it to GPU memory
    float wave_vector[2] = {0,0};

    float *d_wave_vector;
    hipMalloc(&d_wave_vector, 2 * sizeof(*d_wave_vector));
    hipMemcpy(d_wave_vector, wave_vector, 2*sizeof(float), hipMemcpyHostToDevice);

    // Initialize arrays on the GPU to store results per spin system for energy and sum of B2
    thrust::complex<float> *d_store_sum;
    hipMalloc(&d_store_sum, num_iterations*sizeof(*d_store_sum));

    float *d_store_energy;
    hipMalloc(&d_store_energy, num_iterations*sizeof(*d_store_energy));

    //Setup interaction lattice on device
    signed char *d_interactions;
    hipMalloc(&d_interactions, nx*ny*2*sizeof(*d_interactions));

    init_interactions_with_seed(d_interactions, seeds_interactions, nx, ny, p);

    //Synchronize devices
    hipDeviceSynchronize();

    // Loop over number of iterations
    for (int i=0; i<num_iterations; i++){
        
        // Setup black and white lattice arrays on device
        signed char *lattice_b, *lattice_w;
        hipMalloc(&lattice_b, nx * ny/2 * sizeof(*lattice_b));
        hipMalloc(&lattice_w, nx * ny/2 * sizeof(*lattice_w));

        init_spins_with_seed(lattice_b, lattice_w, seeds_spins, nx, ny); 

        // Setup cuRAND generator
        hiprandGenerator_t rng;
        hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
        hiprandSetPseudoRandomGeneratorSeed(rng, seeds_spins);
        float *randvals;
        hipMalloc(&randvals, nx * ny/2 * sizeof(*randvals));
        
        //Synchronize devices
        hipDeviceSynchronize();

        // Warmup iterations
        printf("Starting warmup...\n");

        for (int i = 0; i < nwarmup; i++) {
            update(lattice_b, lattice_w, randvals, rng, d_interactions, inv_temp, nx, ny, coupling_constant);
        }

        //Synchronize devices
        hipDeviceSynchronize();
        
        for (int i = 0; i < niters; i++) {
            update(lattice_b, lattice_w, randvals, rng, d_interactions, inv_temp, nx, ny,coupling_constant);
            if (i % 1000 == 0) printf("Completed %d/%d iterations...\n", i+1, niters);
        }
    
        hipDeviceSynchronize();

        write_lattice(lattice_b, lattice_w, "final.txt", nx, ny);
        write_bonds(d_interactions, "final_bonds.txt" ,nx, ny);
    }
}