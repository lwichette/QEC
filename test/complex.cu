#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand.h>

using namespace std;

const int THREADS = 256;


// Initialize lattice spins
__global__ void init_spins(signed char* lattice, const float* __restrict__ randvals,
    const long long nx, const long long ny) {
        const long long  tid = static_cast<long long>(blockDim.x) * blockIdx.x + threadIdx.x;
        if (tid >= nx * ny) return;
        
        float randval = randvals[tid];
        signed char val = (randval < 0.5f) ? -1 : 1;
        lattice[tid] = val;
}


int main(void){
    // Lattice size, probability, factors,...
    int nx = 10;
    int ny = 10;

    int blocks =(nx * ny * 2 + THREADS - 1) / THREADS;
    
    int num_lattices = 10;

    
    // Setup cuRAND generator
    hiprandGenerator_t rng;
    hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
    hiprandSetPseudoRandomGeneratorSeed(rng, seed);
    float *randvals;
    hipMalloc(&randvals, nx * ny * sizeof(*randvals));
    hiprandGenerateUniform(rng, randvals, nx*ny);
    
    // Initialize lattice
    signed char *h_lattice = (signed char *)malloc(nx*ny*sizeof(signed char));
    signed char *d_lattice;
    hipMalloc(&d_lattice, nx * ny * sizeof(*d_lattice));
    init_spins<<<blocks, THREADS>>>(d_lattice, randvals, nx, ny);

    hipMemcpy(h_lattice, d_lattice, nx*ny*sizeof(signed char), hipMemcpyDeviceToHost);

    // Check if they are the same
    float *randvals1;
    hipMalloc(&randvals1, nx * ny * sizeof(*randvals));
    hiprandGenerateUniform(rng, randvals1, nx*ny);
    
    // Initialize lattice
    signed char *h_lattice1 = (signed char *)malloc(nx*ny*sizeof(signed char));
    signed char *d_lattice1;
    hipMalloc(&d_lattice1, nx * ny * sizeof(*d_lattice1));
    init_spins<<<blocks, THREADS>>>(d_lattice1, randvals1, nx, ny);

    hipMemcpy(h_lattice1, d_lattice1, nx*ny*sizeof(signed char), hipMemcpyDeviceToHost);

    int equal = 0;

    for (int i=0; i<nx*ny;i++){
        if (h_lattice[i] == h_lattice1[i]){
            equal += 1;
        }
    }

    printf("%d", equal);
}